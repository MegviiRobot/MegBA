#include "hip/hip_runtime.h"
/**
* MegBA is Licensed under the Apache License, Version 2.0 (the "License")
*
* Copyright (c) 2021 Megvii Inc. All rights reserved.
*
**/

#include "algo/lm_algo.h"
#include <thrust/device_ptr.h>
#include <thrust/inner_product.h>
#include <thrust/async/reduce.h>
#include <iostream>
#include "linear_system/LM_linear_system.h"
#include "wrapper.hpp"
#include "macro.h"

namespace MegBA {
namespace {
template <typename T>
double computeResidualNorm(const JVD<T> &JV) {
  double residualNormNew = 0.;
  std::vector<std::vector<T>> residualNormNewInFlight;
  residualNormNewInFlight.resize(MemoryPool::getWorldSize());
  for (auto &vec : residualNormNewInFlight) vec.resize(JV.size());
  const auto &cublasHandle = HandleManager::getCUBLASHandle();
  for (int i = 0; i < JV.size(); ++i) {
    for (int j = 0; j < MemoryPool::getWorldSize(); ++j) {
      hipSetDevice(j);
      const T *resPtr = JV(i).getCUDAResPtr()[j];
      Wrapper::cublasGdot::call(cublasHandle[j], MemoryPool::getItemNum(j),
                                resPtr, 1, resPtr, 1,
                                &residualNormNewInFlight[j][i]);
    }
  }
  for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
    hipStream_t stream;
    hipSetDevice(i);
    hipblasGetStream(cublasHandle[i], &stream);
    hipStreamSynchronize(stream);
    for (const auto residualNormNewLanded : residualNormNewInFlight[i]) {
      residualNormNew += residualNormNewLanded;
    }
  }
  return residualNormNew;
}

template <typename T>
inline T l2NormPow2(const T *vector, const std::size_t size) {
  return thrust::inner_product(thrust::device_ptr<const T>(vector),
                               thrust::device_ptr<const T>{vector + size},
                               thrust::device_ptr<const T>(vector), T(0.));
}

template <typename T>
__global__ void JdxpF(const T *grad, const T *deltaX, const T *res,
                      const int *absCameraPosition, const int *absPointPosition,
                      const int nItem, const int cameraDim, const int cameraNum,
                      const int pointDim, T *out) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= nItem) return;
  T sum{0};
  const int absCameraPositionLocal = absCameraPosition[tid];
  const int absPointPositionLocal = absPointPosition[tid];
  for (int i = 0; i < cameraDim; ++i) {
    sum +=
        grad[tid + i * nItem] * deltaX[i + absCameraPositionLocal * cameraDim];
  }
  for (int i = 0; i < pointDim; ++i) {
    sum += grad[tid + (i + cameraDim) * nItem] *
           deltaX[i + cameraDim * cameraNum + absPointPositionLocal * pointDim];
  }
  out[tid] = (sum + res[tid]) * (sum + res[tid]);
}

template <typename T>
double computeRhoDenominator(const JVD<T> &JV, const BaseLinearSystem<T> &linearSystem, const EdgeVector<T> &edges) {
  T rhoDenominator{0};
  std::vector<std::vector<T *>> Jdx;
  Jdx.resize(MemoryPool::getWorldSize());
  const int cameraDim = linearSystem.dim[0];
  const int cameraNum = linearSystem.num[0];
  const int pointDim = linearSystem.dim[1];

  std::vector<std::vector<thrust::system::cuda::unique_eager_future<T>>>
      futures;
  futures.resize(MemoryPool::getWorldSize());

  for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
    hipSetDevice(i);
    const auto nItem = MemoryPool::getItemNum(i);
    const auto &positionContainer = edges.getPositionContainers()[i];
    futures[i].resize(JV.size());
    for (int j = 0; j < JV.size(); ++j) {
      auto &J = JV(j);
      T *ptr;
      MemoryPool::allocateNormal(reinterpret_cast<void **>(&ptr),
                                 nItem * sizeof(T), i);
      dim3 block(std::min((std::size_t)256, nItem));
      dim3 grid((nItem - 1) / block.x + 1);
      ASSERT_CUDA_NO_ERROR();
      JdxpF<<<grid, block>>>(J.getCUDAGradPtr()[i], linearSystem.deltaXPtr[i],
                             J.getCUDAResPtr()[i],
                             positionContainer.absolutePosition[0],
                             positionContainer.absolutePosition[1], nItem,
                             cameraDim, cameraNum, pointDim, ptr);
      ASSERT_CUDA_NO_ERROR();
      futures[i][j] = thrust::async::reduce(
          thrust::cuda::par.on(nullptr), thrust::device_ptr<T>{ptr},
          thrust::device_ptr<T>{ptr} + nItem, T(0.), thrust::plus<T>{});
      Jdx[i].push_back(ptr);
    }
  }
  for (int i = 0; i < futures.size(); ++i) {
    for (int j = futures[i].size() - 1; j >= 0; --j) {
      rhoDenominator += futures[i][j].get();
      MemoryPool::deallocateNormal(reinterpret_cast<void *>(Jdx[i][j]), i);
    }
  }
  return rhoDenominator;
}

template <typename T>
inline T linfNorm(const T *vector, const std::size_t size) {
  return std::abs(
      *thrust::max_element(thrust::device_ptr<const T>{vector},
                           thrust::device_ptr<const T>{vector + size},
                           [] __device__ __host__ (T lhs, T rhs) {
                             return std::abs(lhs) < std::abs(rhs);
                           }));
}
}
template <typename T>
void LMAlgo<T>::solveCUDA(const BaseLinearSystem<T> &baseLinearSystem,
                          const EdgeVector<T> &edges,
                          T *xPtr) {
  const auto &linearSystem = dynamic_cast<const LMLinearSystem<T> &>(baseLinearSystem);
  JVD<T> jvBackup;
  jvBackup = edges.forward();
  ASSERT_CUDA_NO_ERROR();
  edges.buildLinearSystem(jvBackup, linearSystem);
  double residualNorm, residualNormNew = computeResidualNorm(jvBackup);
  std::cout << "start with error: " << residualNormNew / 2
            << ", log error: " << std::log10(residualNormNew / 2) << std::endl;

  MemoryPool::redistribute();
  bool stop{false};
  int k = 0;
  double v = 2.;
  while (!stop && k < this->algoOption.algoOptionLM.maxIter) {
    k++;
    linearSystem.processDiag(this->algoStatus.algoStatusLM);
    linearSystem.solve();
    MemoryPool::redistribute();
    hipSetDevice(0);
    double deltaXL2 = std::sqrt(l2NormPow2(linearSystem.deltaXPtr[0], linearSystem.getHessianShape()));;
    double xL2 = std::sqrt(l2NormPow2(xPtr, linearSystem.getHessianShape()));
    if (deltaXL2 <= this->algoOption.algoOptionLM.epsilon2 * (xL2 + this->algoOption.algoOptionLM.epsilon1)) {
      break;
    }
    edges.update(linearSystem);
    double rhoDenominator = computeRhoDenominator(jvBackup, linearSystem, edges) - residualNormNew;
    residualNorm = residualNormNew;
    JVD<T> jv = edges.forward();
    residualNormNew = computeResidualNorm(jv);
    double rho = -(residualNorm - residualNormNew) / rhoDenominator;
    if (residualNorm > residualNormNew) {
      for (int i = 0; i < jv.size(); ++i) {
        jvBackup(i) = jv(i);
      }
      edges.buildLinearSystem(jv, linearSystem);
      std::cout << k << "-th iter error: " << residualNormNew / 2
                << ", log error: " << std::log10(residualNormNew / 2)
                << std::endl;
      linearSystem.backup();
      edges.backup();
      linearSystem.applyUpdate(xPtr);

      residualNorm = residualNormNew;
      this->algoStatus.algoStatusLM.region /= std::max(1. / 3., 1 - std::pow(2 * rho - 1, 3));
      v = 2.;
      this->algoStatus.algoStatusLM.recoverDiag = false;

      hipSetDevice(0);
      const auto norm =
          linfNorm(linearSystem.g[0], linearSystem.getHessianShape());
      stop = norm <= this->algoOption.algoOptionLM.epsilon1;
    } else {
      linearSystem.rollback();
      edges.rollback();
      residualNormNew = residualNorm;
      this->algoStatus.algoStatusLM.region /= v;
      v *= 2;
      this->algoStatus.algoStatusLM.recoverDiag = true;
    }
  }
}

template class LMAlgo<double>;
template class LMAlgo<float>;
}
