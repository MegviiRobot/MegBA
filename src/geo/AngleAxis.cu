#include "hip/hip_runtime.h"
/**
* MegBA is Licensed under the Apache License, Version 2.0 (the "License")
*
* Copyright (c) 2021 Megvii Inc. All rights reserved.
*
**/

#include "geo/Geo.cuh"
#include "Wrapper.hpp"
#include <Macro.h>

namespace MegBA {
namespace geo {
namespace {
template <typename T>
__global__ void AngleAxisToRotationKernel(
    const int nElm, const int N, const T *da_ptr0, const T *da_ptr1,
    const T *da_ptr2, const T *dv_ptr0, const T *dv_ptr1, const T *dv_ptr2,
    T *R0, T *R1, T *R2, T *R3, T *R4, T *R5, T *R6, T *R7, T *R8, T *dvptr_R0,
    T *dvptr_R1, T *dvptr_R2, T *dvptr_R3, T *dvptr_R4, T *dvptr_R5,
    T *dvptr_R6, T *dvptr_R7, T *dvptr_R8) {
  unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= nElm)
    return;
  const T angle_axis_x = da_ptr0[idx];
  const T angle_axis_y = da_ptr1[idx];
  const T angle_axis_z = da_ptr2[idx];

  const T theta2 = angle_axis_x * angle_axis_x + angle_axis_y * angle_axis_y +
                   angle_axis_z * angle_axis_z;
  if (theta2 > std::numeric_limits<T>::epsilon()) {
    const T theta = Wrapper::sqrtG<T>::call(theta2); // sqrt double
    const T wx = angle_axis_x / theta;
    const T wy = angle_axis_y / theta;
    const T wz = angle_axis_z / theta;

    T sintheta, costheta;

    Wrapper::sincosG<T>::call(theta, &sintheta, &costheta);
    const T one_minor_costheta = T(1.0) - costheta;
    const T wx_mul_one_minor_costheta = wx * one_minor_costheta;
    const T wy_mul_one_minor_costheta = wy * one_minor_costheta;
    const T wz_mul_one_minor_costheta = wz * one_minor_costheta;
    const T wx_mul_wy_mul_one_minor_costheta = wy * wx_mul_one_minor_costheta;
    const T wx_mul_wz_mul_one_minor_costheta = wz * wx_mul_one_minor_costheta;
    const T wy_mul_wz_mul_one_minor_costheta = wz * wy_mul_one_minor_costheta;
    const T wx_mul_sintheta = wx * sintheta;
    const T wy_mul_sintheta = wy * sintheta;
    const T wz_mul_sintheta = wz * sintheta;

    // clang-format on
    const T reciprocal_theta = 1 / theta;
    const T tmp1 = sintheta * reciprocal_theta;
    const T tmpwx = tmp1 * (wx * wx - T(1.0));
    const T tmpwy = tmp1 * (wy * wy - T(1.0));
    const T tmpwz = tmp1 * (wz * wz - T(1.0));

    for (int i = 0; i < N; ++i) {
      unsigned int index = idx + i * nElm;
      const T dv_angle_axis_x = dv_ptr0[index];
      const T dv_angle_axis_y = dv_ptr1[index];
      const T dv_angle_axis_z = dv_ptr2[index];

      const T dv_tmp1 =
          (angle_axis_x * dv_angle_axis_x + angle_axis_y * dv_angle_axis_y +
           angle_axis_z * dv_angle_axis_z);
      const T dv_theta = reciprocal_theta * dv_tmp1;

      const T dv_wx =
          reciprocal_theta *
          (dv_angle_axis_x - angle_axis_x * reciprocal_theta * dv_theta);
      const T dv_wy =
          reciprocal_theta *
          (dv_angle_axis_y - angle_axis_y * reciprocal_theta * dv_theta);
      const T dv_wz =
          reciprocal_theta *
          (dv_angle_axis_z - angle_axis_z * reciprocal_theta * dv_theta);

      dvptr_R0[index] = tmpwx * dv_tmp1 + 2 * wx_mul_one_minor_costheta * dv_wx;
      dvptr_R4[index] = tmpwy * dv_tmp1 + 2 * wy_mul_one_minor_costheta * dv_wy;
      dvptr_R8[index] = tmpwz * dv_tmp1 + 2 * wz_mul_one_minor_costheta * dv_wz;

      dvptr_R1[index] = (wz * costheta + wx * wy_mul_sintheta) * dv_theta +
                        sintheta * dv_wz + wy_mul_one_minor_costheta * dv_wx +
                        wx_mul_one_minor_costheta * dv_wy;

      dvptr_R5[index] = (wx * costheta + wy * wz_mul_sintheta) * dv_theta +
                        sintheta * dv_wx + wz_mul_one_minor_costheta * dv_wy +
                        wy_mul_one_minor_costheta * dv_wz;

      dvptr_R6[index] = (wy * costheta + wx * wz_mul_sintheta) * dv_theta +
                        sintheta * dv_wy + wz_mul_one_minor_costheta * dv_wx +
                        wx_mul_one_minor_costheta * dv_wz;

      dvptr_R2[index] = (-wy * costheta + wx * wz_mul_sintheta) * dv_theta -
                        sintheta * dv_wy + wz_mul_one_minor_costheta * dv_wx +
                        wx_mul_one_minor_costheta * dv_wz;

      dvptr_R3[index] = (-wz * costheta + wx * wy_mul_sintheta) * dv_theta -
                        sintheta * dv_wz + wy_mul_one_minor_costheta * dv_wx +
                        wx_mul_one_minor_costheta * dv_wy;

      dvptr_R7[index] = (-wx * costheta + wy * wz_mul_sintheta) * dv_theta -
                        sintheta * dv_wx + wz_mul_one_minor_costheta * dv_wy +
                        wy_mul_one_minor_costheta * dv_wz;
    }

    R0[idx] = costheta + wx * wx_mul_one_minor_costheta;
    R1[idx] = wz_mul_sintheta + wx_mul_wy_mul_one_minor_costheta;
    R2[idx] = -wy_mul_sintheta + wx_mul_wz_mul_one_minor_costheta;

    R3[idx] = -wz_mul_sintheta + wx_mul_wy_mul_one_minor_costheta;
    R4[idx] = costheta + wy * wy_mul_one_minor_costheta;
    R5[idx] = wx_mul_sintheta + wy_mul_wz_mul_one_minor_costheta;

    R6[idx] = wy_mul_sintheta + wx_mul_wz_mul_one_minor_costheta;
    R7[idx] = -wx_mul_sintheta + wy_mul_wz_mul_one_minor_costheta;
    R8[idx] = costheta + wz * wz_mul_one_minor_costheta;
  } else {
    // Near zero, we switch to using the first order Taylor expansion.
    for (int i = 0; i < N; ++i) {
      unsigned int index = idx + i * nElm;
      const T dv_angle_axis_x = dv_ptr0[index];
      const T dv_angle_axis_y = dv_ptr1[index];
      const T dv_angle_axis_z = dv_ptr2[index];
      dvptr_R0[index] = 0;
      dvptr_R1[index] = dv_angle_axis_z;
      dvptr_R2[index] = -dv_angle_axis_y;
      dvptr_R3[index] = -dv_angle_axis_z;
      dvptr_R4[index] = 0;
      dvptr_R5[index] = dv_angle_axis_x;
      dvptr_R6[index] = dv_angle_axis_y;
      dvptr_R7[index] = -dv_angle_axis_x;
      dvptr_R8[index] = 0;
    }
    R0[idx] = T(1.0);
    R1[idx] = angle_axis_z;
    R2[idx] = -angle_axis_y;

    R3[idx] = -angle_axis_z;
    R4[idx] = T(1.0);
    R5[idx] = angle_axis_x;

    R6[idx] = angle_axis_y;
    R7[idx] = -angle_axis_x;
    R8[idx] = T(1.0);
  }
}

template <typename T>
__global__ void AngleAxisToRotationKernelFastGradKernel(
    const int nElm, const int N, const T *da_ptr0, const T *da_ptr1,
    const T *da_ptr2, const int grad_position0, const int grad_position1,
    const int grad_position2, T *R0, T *R1, T *R2, T *R3, T *R4, T *R5, T *R6,
    T *R7, T *R8, T *dvptr_R0, T *dvptr_R1, T *dvptr_R2, T *dvptr_R3,
    T *dvptr_R4, T *dvptr_R5, T *dvptr_R6, T *dvptr_R7, T *dvptr_R8) {
  unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= nElm)
    return;
  const T angle_axis_x = da_ptr0[idx];
  const T angle_axis_y = da_ptr1[idx];
  const T angle_axis_z = da_ptr2[idx];

  const T theta2 = angle_axis_x * angle_axis_x + angle_axis_y * angle_axis_y +
                   angle_axis_z * angle_axis_z;
  if (theta2 > std::numeric_limits<T>::epsilon()) {
    const T theta = Wrapper::sqrtG<T>::call(theta2); // sqrt double
    const T wx = angle_axis_x / theta;
    const T wy = angle_axis_y / theta;
    const T wz = angle_axis_z / theta;

    T sintheta, costheta;

    Wrapper::sincosG<T>::call(theta, &sintheta, &costheta);
    const T one_minor_costheta = T(1.0) - costheta;
    const T wx_mul_one_minor_costheta = wx * one_minor_costheta;
    const T wy_mul_one_minor_costheta = wy * one_minor_costheta;
    const T wz_mul_one_minor_costheta = wz * one_minor_costheta;
    const T wx_mul_wy_mul_one_minor_costheta = wy * wx_mul_one_minor_costheta;
    const T wx_mul_wz_mul_one_minor_costheta = wz * wx_mul_one_minor_costheta;
    const T wy_mul_wz_mul_one_minor_costheta = wz * wy_mul_one_minor_costheta;
    const T wx_mul_sintheta = wx * sintheta;
    const T wy_mul_sintheta = wy * sintheta;
    const T wz_mul_sintheta = wz * sintheta;

    const T reciprocal_theta = 1 / theta;
    const T tmp1 = sintheta * reciprocal_theta;
    const T tmpwx = tmp1 * (wx * wx - T(1.0));
    const T tmpwy = tmp1 * (wy * wy - T(1.0));
    const T tmpwz = tmp1 * (wz * wz - T(1.0));

    for (int i = 0; i < N; ++i) {
      unsigned int index = idx + i * nElm;
      const T dv_angle_axis_x = i == grad_position0 ? 1 : 0;
      const T dv_angle_axis_y = i == grad_position1 ? 1 : 0;
      const T dv_angle_axis_z = i == grad_position2 ? 1 : 0;

      const T dv_tmp1 =
          (angle_axis_x * dv_angle_axis_x + angle_axis_y * dv_angle_axis_y +
           angle_axis_z * dv_angle_axis_z);
      const T dv_theta = reciprocal_theta * dv_tmp1;

      const T dv_wx =
          reciprocal_theta *
          (dv_angle_axis_x - angle_axis_x * reciprocal_theta * dv_theta);
      const T dv_wy =
          reciprocal_theta *
          (dv_angle_axis_y - angle_axis_y * reciprocal_theta * dv_theta);
      const T dv_wz =
          reciprocal_theta *
          (dv_angle_axis_z - angle_axis_z * reciprocal_theta * dv_theta);

      dvptr_R0[index] = tmpwx * dv_tmp1 + 2 * wx_mul_one_minor_costheta * dv_wx;
      dvptr_R4[index] = tmpwy * dv_tmp1 + 2 * wy_mul_one_minor_costheta * dv_wy;
      dvptr_R8[index] = tmpwz * dv_tmp1 + 2 * wz_mul_one_minor_costheta * dv_wz;

      dvptr_R1[index] = (wz * costheta + wx * wy_mul_sintheta) * dv_theta +
                        sintheta * dv_wz + wy_mul_one_minor_costheta * dv_wx +
                        wx_mul_one_minor_costheta * dv_wy;

      dvptr_R5[index] = (wx * costheta + wy * wz_mul_sintheta) * dv_theta +
                        sintheta * dv_wx + wz_mul_one_minor_costheta * dv_wy +
                        wy_mul_one_minor_costheta * dv_wz;

      dvptr_R6[index] = (wy * costheta + wx * wz_mul_sintheta) * dv_theta +
                        sintheta * dv_wy + wz_mul_one_minor_costheta * dv_wx +
                        wx_mul_one_minor_costheta * dv_wz;

      dvptr_R2[index] = (-wy * costheta + wx * wz_mul_sintheta) * dv_theta -
                        sintheta * dv_wy + wz_mul_one_minor_costheta * dv_wx +
                        wx_mul_one_minor_costheta * dv_wz;

      dvptr_R3[index] = (-wz * costheta + wx * wy_mul_sintheta) * dv_theta -
                        sintheta * dv_wz + wy_mul_one_minor_costheta * dv_wx +
                        wx_mul_one_minor_costheta * dv_wy;

      dvptr_R7[index] = (-wx * costheta + wy * wz_mul_sintheta) * dv_theta -
                        sintheta * dv_wx + wz_mul_one_minor_costheta * dv_wy +
                        wy_mul_one_minor_costheta * dv_wz;
    }

    R0[idx] = costheta + wx * wx_mul_one_minor_costheta;
    R1[idx] = wz_mul_sintheta + wx_mul_wy_mul_one_minor_costheta;
    R2[idx] = -wy_mul_sintheta + wx_mul_wz_mul_one_minor_costheta;

    R3[idx] = -wz_mul_sintheta + wx_mul_wy_mul_one_minor_costheta;
    R4[idx] = costheta + wy * wy_mul_one_minor_costheta;
    R5[idx] = wx_mul_sintheta + wy_mul_wz_mul_one_minor_costheta;

    R6[idx] = wy_mul_sintheta + wx_mul_wz_mul_one_minor_costheta;
    R7[idx] = -wx_mul_sintheta + wy_mul_wz_mul_one_minor_costheta;
    R8[idx] = costheta + wz * wz_mul_one_minor_costheta;
  } else {
    // Near zero, we switch to using the first order Taylor expansion.
    for (int i = 0; i < N; ++i) {
      unsigned int index = idx + i * nElm;
      const T dv_angle_axis_x = i == grad_position0 ? 1 : 0;
      const T dv_angle_axis_y = i == grad_position1 ? 1 : 0;
      const T dv_angle_axis_z = i == grad_position2 ? 1 : 0;
      dvptr_R0[index] = 0;
      dvptr_R1[index] = dv_angle_axis_z;
      dvptr_R2[index] = -dv_angle_axis_y;
      dvptr_R3[index] = -dv_angle_axis_z;
      dvptr_R4[index] = 0;
      dvptr_R5[index] = dv_angle_axis_x;
      dvptr_R6[index] = dv_angle_axis_y;
      dvptr_R7[index] = -dv_angle_axis_x;
      dvptr_R8[index] = 0;
    }
    R0[idx] = T(1.0);
    R1[idx] = angle_axis_z;
    R2[idx] = -angle_axis_y;

    R3[idx] = -angle_axis_z;
    R4[idx] = T(1.0);
    R5[idx] = angle_axis_x;

    R6[idx] = angle_axis_y;
    R7[idx] = -angle_axis_x;
    R8[idx] = T(1.0);
  }
}
}

        template <typename T>
JM33<T> AngleAxisToRotationKernelMatrix(const JV3<T> &AxisAngle) {
  JM33<T> R{};
  const MegBA::JetVector<T> &JV_Template = AxisAngle(0, 0);
  for (int i = 0; i < 3; ++i) {
    for (int j = 0; j < 3; ++j) {
      R(i, j).InitAs(JV_Template);
    }
  }

  bool use_fast_grad{true};
  for (int i = 0; i < 3; ++i)
    use_fast_grad &= AxisAngle(i).get_Grad_Position() != -1;

  const auto N = JV_Template.getGradShape();
  for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
    hipSetDevice(i);
    const auto nElm = JV_Template.get_Elm_Num(i);
    // 512 instead of 1024 for the limitation of registers
    dim3 block_dim(std::min(decltype(nElm)(512), nElm));
    dim3 grid_dim((nElm - 1) / block_dim.x + 1);
    ASSERT_CUDA_NO_ERROR();

    if (use_fast_grad)
      AngleAxisToRotationKernelFastGradKernel<T><<<grid_dim, block_dim>>>(
          nElm, N, AxisAngle(0).get_CUDA_Res_ptr()[i],
          AxisAngle(1).get_CUDA_Res_ptr()[i],
          AxisAngle(2).get_CUDA_Res_ptr()[i], AxisAngle(0).get_Grad_Position(),
          AxisAngle(1).get_Grad_Position(), AxisAngle(2).get_Grad_Position(),
          R(0, 0).get_CUDA_Res_ptr()[i], R(1, 0).get_CUDA_Res_ptr()[i],
          R(2, 0).get_CUDA_Res_ptr()[i], R(0, 1).get_CUDA_Res_ptr()[i],
          R(1, 1).get_CUDA_Res_ptr()[i], R(2, 1).get_CUDA_Res_ptr()[i],
          R(0, 2).get_CUDA_Res_ptr()[i], R(1, 2).get_CUDA_Res_ptr()[i],
          R(2, 2).get_CUDA_Res_ptr()[i], R(0, 0).get_CUDA_Grad_ptr()[i],
          R(1, 0).get_CUDA_Grad_ptr()[i], R(2, 0).get_CUDA_Grad_ptr()[i],
          R(0, 1).get_CUDA_Grad_ptr()[i], R(1, 1).get_CUDA_Grad_ptr()[i],
          R(2, 1).get_CUDA_Grad_ptr()[i], R(0, 2).get_CUDA_Grad_ptr()[i],
          R(1, 2).get_CUDA_Grad_ptr()[i], R(2, 2).get_CUDA_Grad_ptr()[i]);
    else
      AngleAxisToRotationKernel<T><<<grid_dim, block_dim>>>(
          nElm, N, AxisAngle(0, 0).get_CUDA_Res_ptr()[i],
          AxisAngle(1, 0).get_CUDA_Res_ptr()[i],
          AxisAngle(2, 0).get_CUDA_Res_ptr()[i],
          AxisAngle(0, 0).get_CUDA_Grad_ptr()[i],
          AxisAngle(1, 0).get_CUDA_Grad_ptr()[i],
          AxisAngle(2, 0).get_CUDA_Grad_ptr()[i], R(0, 0).get_CUDA_Res_ptr()[i],
          R(1, 0).get_CUDA_Res_ptr()[i], R(2, 0).get_CUDA_Res_ptr()[i],
          R(0, 1).get_CUDA_Res_ptr()[i], R(1, 1).get_CUDA_Res_ptr()[i],
          R(2, 1).get_CUDA_Res_ptr()[i], R(0, 2).get_CUDA_Res_ptr()[i],
          R(1, 2).get_CUDA_Res_ptr()[i], R(2, 2).get_CUDA_Res_ptr()[i],
          R(0, 0).get_CUDA_Grad_ptr()[i], R(1, 0).get_CUDA_Grad_ptr()[i],
          R(2, 0).get_CUDA_Grad_ptr()[i], R(0, 1).get_CUDA_Grad_ptr()[i],
          R(1, 1).get_CUDA_Grad_ptr()[i], R(2, 1).get_CUDA_Grad_ptr()[i],
          R(0, 2).get_CUDA_Grad_ptr()[i], R(1, 2).get_CUDA_Grad_ptr()[i],
          R(2, 2).get_CUDA_Grad_ptr()[i]);
    ASSERT_CUDA_NO_ERROR();
  }

  return R;
}

template <typename T>
JM33<T>
AngleAxisToRotationKernelMatrix(const Eigen::Map<const JVD<T>> &AxisAngle) {
  JM33<T> R{};
  const MegBA::JetVector<T> &JV_Template = AxisAngle(0, 0);
  for (int i = 0; i < 3; ++i) {
    for (int j = 0; j < 3; ++j) {
      R(i, j).InitAs(JV_Template);
    }
  }

  bool use_fast_grad{true};
  for (int i = 0; i < 3; ++i)
    use_fast_grad &= AxisAngle(i).get_Grad_Position() != -1;

  const auto N = JV_Template.getGradShape();
  for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
    hipSetDevice(i);
    const auto nElm = JV_Template.get_Elm_Num(i);
    // 512 instead of 1024 for the limitation of registers
    dim3 block_dim(std::min(decltype(nElm)(512), nElm));
    dim3 grid_dim((nElm - 1) / block_dim.x + 1);
    ASSERT_CUDA_NO_ERROR();

    if (use_fast_grad)
      AngleAxisToRotationKernelFastGradKernel<T><<<grid_dim, block_dim>>>(
          nElm, N, AxisAngle(0).get_CUDA_Res_ptr()[i],
          AxisAngle(1).get_CUDA_Res_ptr()[i],
          AxisAngle(2).get_CUDA_Res_ptr()[i], AxisAngle(0).get_Grad_Position(),
          AxisAngle(1).get_Grad_Position(), AxisAngle(2).get_Grad_Position(),
          R(0, 0).get_CUDA_Res_ptr()[i], R(1, 0).get_CUDA_Res_ptr()[i],
          R(2, 0).get_CUDA_Res_ptr()[i], R(0, 1).get_CUDA_Res_ptr()[i],
          R(1, 1).get_CUDA_Res_ptr()[i], R(2, 1).get_CUDA_Res_ptr()[i],
          R(0, 2).get_CUDA_Res_ptr()[i], R(1, 2).get_CUDA_Res_ptr()[i],
          R(2, 2).get_CUDA_Res_ptr()[i], R(0, 0).get_CUDA_Grad_ptr()[i],
          R(1, 0).get_CUDA_Grad_ptr()[i], R(2, 0).get_CUDA_Grad_ptr()[i],
          R(0, 1).get_CUDA_Grad_ptr()[i], R(1, 1).get_CUDA_Grad_ptr()[i],
          R(2, 1).get_CUDA_Grad_ptr()[i], R(0, 2).get_CUDA_Grad_ptr()[i],
          R(1, 2).get_CUDA_Grad_ptr()[i], R(2, 2).get_CUDA_Grad_ptr()[i]);
    else
      AngleAxisToRotationKernel<T><<<grid_dim, block_dim>>>(
          nElm, N, AxisAngle(0, 0).get_CUDA_Res_ptr()[i],
          AxisAngle(1, 0).get_CUDA_Res_ptr()[i],
          AxisAngle(2, 0).get_CUDA_Res_ptr()[i],
          AxisAngle(0, 0).get_CUDA_Grad_ptr()[i],
          AxisAngle(1, 0).get_CUDA_Grad_ptr()[i],
          AxisAngle(2, 0).get_CUDA_Grad_ptr()[i], R(0, 0).get_CUDA_Res_ptr()[i],
          R(1, 0).get_CUDA_Res_ptr()[i], R(2, 0).get_CUDA_Res_ptr()[i],
          R(0, 1).get_CUDA_Res_ptr()[i], R(1, 1).get_CUDA_Res_ptr()[i],
          R(2, 1).get_CUDA_Res_ptr()[i], R(0, 2).get_CUDA_Res_ptr()[i],
          R(1, 2).get_CUDA_Res_ptr()[i], R(2, 2).get_CUDA_Res_ptr()[i],
          R(0, 0).get_CUDA_Grad_ptr()[i], R(1, 0).get_CUDA_Grad_ptr()[i],
          R(2, 0).get_CUDA_Grad_ptr()[i], R(0, 1).get_CUDA_Grad_ptr()[i],
          R(1, 1).get_CUDA_Grad_ptr()[i], R(2, 1).get_CUDA_Grad_ptr()[i],
          R(0, 2).get_CUDA_Grad_ptr()[i], R(1, 2).get_CUDA_Grad_ptr()[i],
          R(2, 2).get_CUDA_Grad_ptr()[i]);
    ASSERT_CUDA_NO_ERROR();
  }

  return R;
}

template JM33<float>
AngleAxisToRotationKernelMatrix(const JV3<float> &AxisAngle);
template JM33<double>
AngleAxisToRotationKernelMatrix(const JV3<double> &AxisAngle);

template JM33<float>
AngleAxisToRotationKernelMatrix(const Eigen::Map<const JVD<float>> &AxisAngle);
template JM33<double>
AngleAxisToRotationKernelMatrix(const Eigen::Map<const JVD<double>> &AxisAngle);
}
}