#include "hip/hip_runtime.h"
/**
* MegBA is Licensed under the Apache License, Version 2.0 (the "License")
*
* Copyright (c) 2021 Megvii Inc. All rights reserved.
*
**/

#include <geo/Geo.cuh>

namespace MegBA {
namespace geo {
namespace {
template <typename T>
__global__ void RadialDistortionNoGradKernel(
    const int nElm, const int N, const T *px_da_ptr, const T *py_da_ptr,
    const T *px_dv_ptr, const T *py_dv_ptr, const T *f_ptr, const T *k1_ptr,
    const T *k2_ptr, T *da_ptr, T *dv_ptr) {
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= nElm)
    return;
  T f = f_ptr[tid], k1 = k1_ptr[tid], k2 = k2_ptr[tid];

  T px = px_da_ptr[tid];

  T py = py_da_ptr[tid];

  T l2_pow2 = px * px + py * py;

  T partial = 2 * f * (k1 + 2 * k2 * l2_pow2);
  for (unsigned int i = 0; i < N; ++i)
    dv_ptr[tid + nElm * i] = partial * (px_dv_ptr[tid + nElm * i] * px +
                                        py_dv_ptr[tid + nElm * i] * py);

  da_ptr[tid] = f * (T(1.) + k1 * l2_pow2 + k2 * l2_pow2 * l2_pow2);
}

template <typename T>
__global__ void
RadialDistortionKernel(const int nElm, const int N, const T *px_da_ptr,
                       const T *py_da_ptr, const T *px_dv_ptr,
                       const T *py_dv_ptr, const T *f_ptr, const T *k1_ptr,
                       const T *k2_ptr, const T *f_dv_ptr, const T *k1_dv_ptr,
                       const T *k2_dv_ptr, T *da_ptr, T *dv_ptr) {
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= nElm)
    return;
  T f = f_ptr[tid], k1 = k1_ptr[tid], k2 = k2_ptr[tid];

  T px = px_da_ptr[tid];

  T py = py_da_ptr[tid];

  T l2_pow2 = px * px + py * py;

  T partial = 2 * f * (k1 + 2 * k2 * l2_pow2);
  for (unsigned int i = 0; i < N; ++i) {
    unsigned int index = tid + nElm * i;
    dv_ptr[index] =
        partial *
            (px_dv_ptr[tid + nElm * i] * px + py_dv_ptr[tid + nElm * i] * py) +
        f_dv_ptr[index] * (T(1.) + k1 * l2_pow2 + k2 * l2_pow2 * l2_pow2) +
        k1_dv_ptr[index] * f * l2_pow2 +
        k2_dv_ptr[index] * f * l2_pow2 * l2_pow2;
  }

  da_ptr[tid] = f * (T(1.) + k1 * l2_pow2 + k2 * l2_pow2 * l2_pow2);
}

template <typename T>
__global__ void RadialDistortionFastGradKernel(
    const int nElm, const int N, const T *px_da_ptr, const T *py_da_ptr,
    const T *px_dv_ptr, const T *py_dv_ptr, const T *f_ptr, const T *k1_ptr,
    const T *k2_ptr, const int f_grad_position, const int k1_grad_position,
    const int k2_grad_position, T *da_ptr, T *dv_ptr) {
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= nElm)
    return;
  T f = f_ptr[tid], k1 = k1_ptr[tid], k2 = k2_ptr[tid];

  T px = px_da_ptr[tid];

  T py = py_da_ptr[tid];

  T l2_pow2 = px * px + py * py;

  T partial = 2 * f * (k1 + 2 * k2 * l2_pow2);
  for (unsigned int i = 0; i < N; ++i) {
    unsigned int index = tid + nElm * i;
    dv_ptr[index] = partial * (px_dv_ptr[tid + nElm * i] * px +
                               py_dv_ptr[tid + nElm * i] * py) +
                    (i == f_grad_position ? 1 : 0) *
                        (T(1.) + k1 * l2_pow2 + k2 * l2_pow2 * l2_pow2) +
                    (i == k1_grad_position ? 1 : 0) * f * l2_pow2 +
                    (i == k2_grad_position ? 1 : 0) * f * l2_pow2 * l2_pow2;
  }

  da_ptr[tid] = f * (T(1.) + k1 * l2_pow2 + k2 * l2_pow2 * l2_pow2);
}

template <typename T>
void RadialDistortionImpl(const JV3<T> &point, const JV3<T> &intrinsic,
                          JetVector<T> &out) {
  const auto N = out.getGradShape();
  bool use_fast_grad{true};
  for (int i = 0; i < 3; ++i)
    use_fast_grad &= intrinsic(i).get_Grad_Position() != -1;

  for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
    hipSetDevice(i);
    const auto nElm = out.get_Elm_Num(i);
    dim3 block_dim(std::min(decltype(nElm)(256), nElm));
    dim3 grid_dim((nElm - 1) / block_dim.x + 1);
    if (intrinsic(0).getGradShape() == 0) {
      RadialDistortionNoGradKernel<T><<<grid_dim, block_dim>>>(
          nElm, N, point(0).get_CUDA_Res_ptr()[i],
          point(1).get_CUDA_Res_ptr()[i], point(0).get_CUDA_Grad_ptr()[i],
          point(1).get_CUDA_Grad_ptr()[i], intrinsic(0).get_CUDA_Res_ptr()[i],
          intrinsic(1).get_CUDA_Res_ptr()[i],
          intrinsic(2).get_CUDA_Res_ptr()[i], out.get_CUDA_Res_ptr()[i],
          out.get_CUDA_Grad_ptr()[i]);
    } else {
      if (use_fast_grad) {
        RadialDistortionFastGradKernel<T><<<grid_dim, block_dim>>>(
            nElm, N, point(0).get_CUDA_Res_ptr()[i],
            point(1).get_CUDA_Res_ptr()[i], point(0).get_CUDA_Grad_ptr()[i],
            point(1).get_CUDA_Grad_ptr()[i], intrinsic(0).get_CUDA_Res_ptr()[i],
            intrinsic(1).get_CUDA_Res_ptr()[i],
            intrinsic(2).get_CUDA_Res_ptr()[i],
            intrinsic(0).get_Grad_Position(), intrinsic(1).get_Grad_Position(),
            intrinsic(2).get_Grad_Position(), out.get_CUDA_Res_ptr()[i],
            out.get_CUDA_Grad_ptr()[i]);
      } else {
        RadialDistortionKernel<T><<<grid_dim, block_dim>>>(
            nElm, N, point(0).get_CUDA_Res_ptr()[i],
            point(1).get_CUDA_Res_ptr()[i], point(0).get_CUDA_Grad_ptr()[i],
            point(1).get_CUDA_Grad_ptr()[i], intrinsic(0).get_CUDA_Res_ptr()[i],
            intrinsic(1).get_CUDA_Res_ptr()[i],
            intrinsic(2).get_CUDA_Res_ptr()[i],
            intrinsic(0).get_CUDA_Grad_ptr()[i],
            intrinsic(1).get_CUDA_Grad_ptr()[i],
            intrinsic(2).get_CUDA_Grad_ptr()[i], out.get_CUDA_Res_ptr()[i],
            out.get_CUDA_Grad_ptr()[i]);
      }
    }
  }
}

template <typename T>
void RadialDistortionImpl(const JV3<T> &point,
                          const Eigen::Map<const JV3<T>> &intrinsic,
                          JetVector<T> &out) {
  const auto N = out.getGradShape();
  bool use_fast_grad{true};
  for (int i = 0; i < 3; ++i)
    use_fast_grad &= intrinsic(i).get_Grad_Position() != -1;

  for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
    hipSetDevice(i);
    const auto nElm = out.get_Elm_Num(i);
    dim3 block_dim(std::min(decltype(nElm)(256), nElm));
    dim3 grid_dim((nElm - 1) / block_dim.x + 1);
    if (intrinsic(0).getGradShape() == 0) {
      RadialDistortionNoGradKernel<T><<<grid_dim, block_dim>>>(
          nElm, N, point(0).get_CUDA_Res_ptr()[i],
          point(1).get_CUDA_Res_ptr()[i], point(0).get_CUDA_Grad_ptr()[i],
          point(1).get_CUDA_Grad_ptr()[i], intrinsic(0).get_CUDA_Res_ptr()[i],
          intrinsic(1).get_CUDA_Res_ptr()[i],
          intrinsic(2).get_CUDA_Res_ptr()[i], out.get_CUDA_Res_ptr()[i],
          out.get_CUDA_Grad_ptr()[i]);
    } else {
      if (use_fast_grad) {
        RadialDistortionFastGradKernel<T><<<grid_dim, block_dim>>>(
            nElm, N, point(0).get_CUDA_Res_ptr()[i],
            point(1).get_CUDA_Res_ptr()[i], point(0).get_CUDA_Grad_ptr()[i],
            point(1).get_CUDA_Grad_ptr()[i], intrinsic(0).get_CUDA_Res_ptr()[i],
            intrinsic(1).get_CUDA_Res_ptr()[i],
            intrinsic(2).get_CUDA_Res_ptr()[i],
            intrinsic(0).get_Grad_Position(), intrinsic(1).get_Grad_Position(),
            intrinsic(2).get_Grad_Position(), out.get_CUDA_Res_ptr()[i],
            out.get_CUDA_Grad_ptr()[i]);
      } else {
        RadialDistortionKernel<T><<<grid_dim, block_dim>>>(
            nElm, N, point(0).get_CUDA_Res_ptr()[i],
            point(1).get_CUDA_Res_ptr()[i], point(0).get_CUDA_Grad_ptr()[i],
            point(1).get_CUDA_Grad_ptr()[i], intrinsic(0).get_CUDA_Res_ptr()[i],
            intrinsic(1).get_CUDA_Res_ptr()[i],
            intrinsic(2).get_CUDA_Res_ptr()[i],
            intrinsic(0).get_CUDA_Grad_ptr()[i],
            intrinsic(1).get_CUDA_Grad_ptr()[i],
            intrinsic(2).get_CUDA_Grad_ptr()[i], out.get_CUDA_Res_ptr()[i],
            out.get_CUDA_Grad_ptr()[i]);
      }
    }
  }
}

template <typename T>
void RadialDistortionImpl(const JV3<T> &point,
                          const Eigen::Map<const JVD<T>> &intrinsic,
                          JetVector<T> &out) {
  const auto N = out.getGradShape();
  bool use_fast_grad{true};
  for (int i = 0; i < 3; ++i)
    use_fast_grad &= intrinsic(i).get_Grad_Position() != -1;

  for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
    hipSetDevice(i);
    const auto nElm = out.get_Elm_Num(i);
    dim3 block_dim(std::min(decltype(nElm)(256), nElm));
    dim3 grid_dim((nElm - 1) / block_dim.x + 1);
    if (intrinsic(0).getGradShape() == 0) {
      RadialDistortionNoGradKernel<T><<<grid_dim, block_dim>>>(
          nElm, N, point(0).get_CUDA_Res_ptr()[i],
          point(1).get_CUDA_Res_ptr()[i], point(0).get_CUDA_Grad_ptr()[i],
          point(1).get_CUDA_Grad_ptr()[i], intrinsic(0).get_CUDA_Res_ptr()[i],
          intrinsic(1).get_CUDA_Res_ptr()[i],
          intrinsic(2).get_CUDA_Res_ptr()[i], out.get_CUDA_Res_ptr()[i],
          out.get_CUDA_Grad_ptr()[i]);
    } else {
      if (use_fast_grad) {
        RadialDistortionFastGradKernel<T><<<grid_dim, block_dim>>>(
            nElm, N, point(0).get_CUDA_Res_ptr()[i],
            point(1).get_CUDA_Res_ptr()[i], point(0).get_CUDA_Grad_ptr()[i],
            point(1).get_CUDA_Grad_ptr()[i], intrinsic(0).get_CUDA_Res_ptr()[i],
            intrinsic(1).get_CUDA_Res_ptr()[i],
            intrinsic(2).get_CUDA_Res_ptr()[i],
            intrinsic(0).get_Grad_Position(), intrinsic(1).get_Grad_Position(),
            intrinsic(2).get_Grad_Position(), out.get_CUDA_Res_ptr()[i],
            out.get_CUDA_Grad_ptr()[i]);
      } else {
        RadialDistortionKernel<T><<<grid_dim, block_dim>>>(
            nElm, N, point(0).get_CUDA_Res_ptr()[i],
            point(1).get_CUDA_Res_ptr()[i], point(0).get_CUDA_Grad_ptr()[i],
            point(1).get_CUDA_Grad_ptr()[i], intrinsic(0).get_CUDA_Res_ptr()[i],
            intrinsic(1).get_CUDA_Res_ptr()[i],
            intrinsic(2).get_CUDA_Res_ptr()[i],
            intrinsic(0).get_CUDA_Grad_ptr()[i],
            intrinsic(1).get_CUDA_Grad_ptr()[i],
            intrinsic(2).get_CUDA_Grad_ptr()[i], out.get_CUDA_Res_ptr()[i],
            out.get_CUDA_Grad_ptr()[i]);
      }
    }
  }
}
}

template <typename T>
JetVector<T> RadialDistortion(const JV3<T> &point, const JV3<T> &intrinsic) {
  return JetVector<T>{point(0, 0), [&](JetVector<T> &out) {
                         RadialDistortionImpl(point, intrinsic, out);
                       }};
}

template <typename T>
JetVector<T> RadialDistortion(const JV3<T> &point,
                               const Eigen::Map<const JV3<T>> &intrinsic) {
  return JetVector<T>{point(0), [&](JetVector<T> &out) {
                         RadialDistortionImpl(point, intrinsic, out);
                       }};
}

template <typename T>
JetVector<T> RadialDistortion(const JV3<T> &point,
                               const Eigen::Map<const JVD<T>> &intrinsic) {
  assert(intrinsic.rows() == 3 && intrinsic.cols() == 1);
  return JetVector<T>{point(0), [&](JetVector<T> &out) {
                         RadialDistortionImpl(point, intrinsic, out);
                       }};
}

template JetVector<float> RadialDistortion(const JV3<float> &point,
                                            const JV3<float> &intrinsic);
template JetVector<double> RadialDistortion(const JV3<double> &point,
                                             const JV3<double> &intrinsic);
template JetVector<float>
RadialDistortion(const JV3<float> &point,
                 const Eigen::Map<const JV3<float>> &intrinsic);
template JetVector<double>
RadialDistortion(const JV3<double> &point,
                 const Eigen::Map<const JV3<double>> &intrinsic);
template JetVector<float>
RadialDistortion(const JV3<float> &point,
                 const Eigen::Map<const JVD<float>> &intrinsic);
template JetVector<double>
RadialDistortion(const JV3<double> &point,
                 const Eigen::Map<const JVD<double>> &intrinsic);
}
}