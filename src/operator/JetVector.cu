/**
* MegBA is Licensed under the Apache License, Version 2.0 (the "License")
*
* Copyright (c) 2021 Megvii Inc. All rights reserved.
*
**/

#include <common.h>
#include <operator/jet_vector.h>
#include <resource/memory_pool.h>
#include <macro.h>
#include <memory>

namespace MegBA {
template <typename T> void JetVector<T>::initAsCUDA(const JetVector<T> &f) {
  const auto world_size = MemoryPool::getWorldSize();
  std::vector<void *> da_ptr, dv_ptr;
  MemoryPool::allocateJetVector(&da_ptr, &dv_ptr, _N, _nElm, sizeof(T));
  _dvPtr.clear();
  _daPtr.clear();
  _dvPtr.resize(world_size);
  _daPtr.resize(world_size);
  for (int i = 0; i < world_size; ++i) {
    _dvPtr[i] = reinterpret_cast<T *>(dv_ptr[i]);
    _daPtr[i] = reinterpret_cast<T *>(da_ptr[i]);
  }
}

template <typename T> JetVector<T> &JetVector<T>::CUDA() {
  if (!IsEmpty()) {
    auto N = _N;
    auto nElm = _nElm;
    switch (_device) {
    case Device::CUDA: {
      break;
    }
    case Device::CPU: {
      // save counter
      CPU2CUDA(*this);
      clear();
      break;
    }
    }  // switch _device
    _N = N;
    _nElm = nElm;
  }
  _device = Device::CUDA;
  return *this;
}

template <typename T> void JetVector<T>::CUDA2CPU(const JetVector<T> &f) {
  const auto world_size = MemoryPool::getWorldSize();
  _haData.resize(_nElm);
  _hvData.resize(_N);
  for (auto &v : _hvData)
    v.resize(_nElm);

  std::size_t start_idx{0};
  for (int i = 0; i < world_size; ++i) {
    hipSetDevice(i);
    std::size_t nElm{getElmNum(i)};
    hipMemcpyAsync(&_haData[start_idx], f._daPtr[i], nElm * sizeof(T),
                    hipMemcpyDeviceToHost);
    if (_gradPosition == -1) {
      for (unsigned int j = 0; j < _N; ++j)
        hipMemcpyAsync(&_hvData[j][start_idx], &f._dvPtr[i][j * nElm],
                        nElm * sizeof(T), hipMemcpyDeviceToHost);
    }
    start_idx += nElm;
  }
}

template <typename T> void JetVector<T>::CPU2CUDA(const JetVector<T> &f) {
  const auto world_size = MemoryPool::getWorldSize();
  // if _daPtr != nullptr if binded
  if (_daPtr.empty()) {
    if (_pureScalarFlag) {
      _daPtr.resize(world_size);
      std::size_t start_idx{0};
      for (int i = 0; i < world_size; ++i) {
        hipSetDevice(i);
        hipMalloc(&_daPtr[i], _nElm * sizeof(T));
        std::size_t nElm{getElmNum(i)};
        hipMemcpyAsync(_daPtr[i], &f._haData[start_idx], nElm * sizeof(T),
                        hipMemcpyHostToDevice);
        start_idx += nElm;
      }
      return;
    }
    std::vector<void *> da_ptr{}, dv_ptr{};
    MemoryPool::allocateJetVector(&da_ptr, &dv_ptr, _N, _nElm, sizeof(T));
    // _dvPtr must be nullptr
    _dvPtr.clear();
    _dvPtr.reserve(world_size);
    for (int i = 0; i < world_size; ++i)
      _dvPtr.push_back(reinterpret_cast<T *>(dv_ptr[i]));

    _daPtr.clear();
    _daPtr.reserve(world_size);
    for (int i = 0; i < world_size; ++i)
      _daPtr.push_back(reinterpret_cast<T *>(da_ptr[i]));

    std::size_t start_idx{0};
    for (int i = 0; i < world_size; ++i) {
      hipSetDevice(i);
      std::size_t nElm{getElmNum(i)};
      hipMemcpyAsync(_daPtr[i], &f._haData[start_idx], nElm * sizeof(T),
                      hipMemcpyHostToDevice);
      for (unsigned int j = 0; j < _N; ++j)
        hipMemcpyAsync(&_dvPtr[i][j * nElm], &f._hvData[j][start_idx],
                        nElm * sizeof(T), hipMemcpyHostToDevice);
      start_idx += nElm;
    }
  } else {
    std::size_t start_idx{0};
    for (int i = 0; i < world_size; ++i) {
      hipSetDevice(i);
      std::size_t nElm{getElmNum(i)};
      hipMemcpyAsync(_daPtr[i], &f._haData[start_idx], nElm * sizeof(T),
                      hipMemcpyHostToDevice);
      start_idx += nElm;
    }
  }
}

template <typename T> void JetVector<T>::CUDA2CUDA(const JetVector<T> &f) {
  const auto world_size = MemoryPool::getWorldSize();
  if (_daPtr.empty()) {
    std::vector<void *> da_ptr{}, dv_ptr{};
    MemoryPool::allocateJetVector(&da_ptr, &dv_ptr, _N, _nElm, sizeof(T));
    _dvPtr.clear();
    _daPtr.clear();
    _dvPtr.reserve(world_size);
    _daPtr.reserve(world_size);
    for (int i = 0; i < world_size; ++i) {
      _dvPtr.push_back(reinterpret_cast<T *>(dv_ptr[i]));
      _daPtr.push_back(reinterpret_cast<T *>(da_ptr[i]));
    }
  }
  for (int i = 0; i < world_size; ++i) {
    hipSetDevice(i);
    std::size_t nElm{getElmNum(i)};
    hipMemcpyAsync(_daPtr[i], f._daPtr[i], nElm * sizeof(T),
                    hipMemcpyDeviceToDevice);
    hipMemcpyAsync(_dvPtr[i], f._dvPtr[i], _N * nElm * sizeof(T),
                    hipMemcpyDeviceToDevice);
  }
}

template <typename T>
std::ostream &ostreamCUDA(std::ostream &s, const JetVector<T> &z) {
  auto N = z.getGradShape();
  auto nElm = z.getElmNum();
  std::unique_ptr<T[]> Res{new T[nElm]};
  std::vector<std::unique_ptr<T[]>> Grad;
  Grad.reserve(N);
  for (int i = 0; i < N; ++i)
    Grad.emplace_back(new T[nElm]);
  std::size_t start_idx{0};
  for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
    hipSetDevice(i);
    std::size_t nElm = z.getElmNum(i);
    hipMemcpyAsync(&Res[start_idx], z.getCUDAResPtr()[i], nElm * sizeof(T),
                    hipMemcpyDeviceToHost);
    for (unsigned int j = 0; j < N; ++j)
      hipMemcpyAsync(&Grad[j][start_idx], &z.getCUDAGradPtr()[i][j * nElm],
                      nElm * sizeof(T), hipMemcpyDeviceToHost);
    start_idx += nElm;
  }
  s << "[Res: "
    << "[ ";
  for (std::size_t i = 0; i < nElm; ++i)
    s << Res[i] << ", ";
  s << "]," << std::endl;
  for (unsigned int i = 0; i < N; ++i) {
    s << "Grad[" << i << "]: "
      << "[ ";
    for (std::size_t j = 0; j < nElm; ++j)
      s << Grad[i][j] << ", ";
    s << "]," << std::endl;
  }
  s << "_device: " << std::to_string(z.getDevice()) << "]";
  return s;
}
template std::ostream &ostreamCUDA<float>(std::ostream &s,
                                          const JetVector<float> &);
template std::ostream &ostreamCUDA<double>(std::ostream &s,
                                           const JetVector<double> &);

template class JetVector<float>;
template class JetVector<double>;
}  // namespace MegBA
