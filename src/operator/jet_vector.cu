/**
* MegBA is Licensed under the Apache License, Version 2.0 (the "License")
*
* Copyright (c) 2021 Megvii Inc. All rights reserved.
*
**/

#include "operator/jet_vector.h"
#include <memory>
#include "resource/memory_pool.h"
#include "common.h"

namespace MegBA {
template <typename T> void JetVector<T>::initAsCUDA(const JetVector<T> &f) {
  const auto world_size = MemoryPool::getWorldSize();
  std::vector<void *> da_ptr, dv_ptr;
  MemoryPool::allocateJetVector(&da_ptr, &dv_ptr, _N, _nEle, sizeof(T));
  _dvPtr.clear();
  _daPtr.clear();
  _dvPtr.resize(world_size);
  _daPtr.resize(world_size);
  for (int i = 0; i < world_size; ++i) {
    _dvPtr[i] = reinterpret_cast<T *>(dv_ptr[i]);
    _daPtr[i] = reinterpret_cast<T *>(da_ptr[i]);
  }
}

template <typename T> JetVector<T> &JetVector<T>::CUDA() {
  if (!IsEmpty()) {
    auto N = _N;
    auto nEle = _nEle;
    switch (_device) {
    case Device::CUDA: {
      break;
    }
    case Device::CPU: {
      // save counter
      CPU2CUDA(*this);
      clear();
      break;
    }
    }  // switch _device
    _N = N;
    _nEle = nEle;
  }
  _device = Device::CUDA;
  return *this;
}

template <typename T> void JetVector<T>::CUDA2CPU(const JetVector<T> &f) {
  const auto world_size = MemoryPool::getWorldSize();
  _haData.resize(_nEle);
  _hvData.resize(_N);
  for (auto &v : _hvData)
    v.resize(_nEle);

  std::size_t start_idx{0};
  for (int i = 0; i < world_size; ++i) {
    hipSetDevice(i);
    std::size_t nEle{getEleNum(i)};
    hipMemcpyAsync(&_haData[start_idx], f._daPtr[i], nEle * sizeof(T),
                    hipMemcpyDeviceToHost);
    if (_gradPosition == -1) {
      for (unsigned int j = 0; j < _N; ++j)
        hipMemcpyAsync(&_hvData[j][start_idx], &f._dvPtr[i][j * nEle],
                        nEle * sizeof(T), hipMemcpyDeviceToHost);
    }
    start_idx += nEle;
  }
}

template <typename T> void JetVector<T>::CPU2CUDA(const JetVector<T> &f) {
  const auto world_size = MemoryPool::getWorldSize();
  // if _daPtr != nullptr if binded
  if (_daPtr.empty()) {
    if (_pureScalarFlag) {
      _daPtr.resize(world_size);
      std::size_t start_idx{0};
      for (int i = 0; i < world_size; ++i) {
        hipSetDevice(i);
        hipMalloc(&_daPtr[i], _nEle * sizeof(T));
        std::size_t nEle{getEleNum(i)};
        hipMemcpyAsync(_daPtr[i], &f._haData[start_idx], nEle * sizeof(T),
                        hipMemcpyHostToDevice);
        start_idx += nEle;
      }
      return;
    }
    std::vector<void *> da_ptr{}, dv_ptr{};
    MemoryPool::allocateJetVector(&da_ptr, &dv_ptr, _N, _nEle, sizeof(T));
    // _dvPtr must be nullptr
    _dvPtr.clear();
    _dvPtr.reserve(world_size);
    for (int i = 0; i < world_size; ++i)
      _dvPtr.push_back(reinterpret_cast<T *>(dv_ptr[i]));

    _daPtr.clear();
    _daPtr.reserve(world_size);
    for (int i = 0; i < world_size; ++i)
      _daPtr.push_back(reinterpret_cast<T *>(da_ptr[i]));

    std::size_t start_idx{0};
    for (int i = 0; i < world_size; ++i) {
      hipSetDevice(i);
      std::size_t nEle{getEleNum(i)};
      hipMemcpyAsync(_daPtr[i], &f._haData[start_idx], nEle * sizeof(T),
                      hipMemcpyHostToDevice);
      for (unsigned int j = 0; j < _N; ++j)
        hipMemcpyAsync(&_dvPtr[i][j * nEle], &f._hvData[j][start_idx],
                        nEle * sizeof(T), hipMemcpyHostToDevice);
      start_idx += nEle;
    }
  } else {
    std::size_t start_idx{0};
    for (int i = 0; i < world_size; ++i) {
      hipSetDevice(i);
      std::size_t nEle{getEleNum(i)};
      hipMemcpyAsync(_daPtr[i], &f._haData[start_idx], nEle * sizeof(T),
                      hipMemcpyHostToDevice);
      start_idx += nEle;
    }
  }
}

template <typename T> void JetVector<T>::CUDA2CUDA(const JetVector<T> &f) {
  const auto world_size = MemoryPool::getWorldSize();
  if (_daPtr.empty()) {
    std::vector<void *> da_ptr{}, dv_ptr{};
    MemoryPool::allocateJetVector(&da_ptr, &dv_ptr, _N, _nEle, sizeof(T));
    _dvPtr.clear();
    _daPtr.clear();
    _dvPtr.reserve(world_size);
    _daPtr.reserve(world_size);
    for (int i = 0; i < world_size; ++i) {
      _dvPtr.push_back(reinterpret_cast<T *>(dv_ptr[i]));
      _daPtr.push_back(reinterpret_cast<T *>(da_ptr[i]));
    }
  }
  for (int i = 0; i < world_size; ++i) {
    hipSetDevice(i);
    std::size_t nEle{getEleNum(i)};
    hipMemcpyAsync(_daPtr[i], f._daPtr[i], nEle * sizeof(T),
                    hipMemcpyDeviceToDevice);
    hipMemcpyAsync(_dvPtr[i], f._dvPtr[i], _N * nEle * sizeof(T),
                    hipMemcpyDeviceToDevice);
  }
}

template <typename T>
std::ostream &ostreamCUDA(std::ostream &s, const JetVector<T> &z) {
  auto N = z.getGradShape();
  auto nEle = z.getEleNum();
  std::unique_ptr<T[]> Res{new T[nEle]};
  std::vector<std::unique_ptr<T[]>> Grad;
  Grad.reserve(N);
  for (int i = 0; i < N; ++i)
    Grad.emplace_back(new T[nEle]);
  std::size_t start_idx{0};
  for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
    hipSetDevice(i);
    std::size_t nEle = z.getEleNum(i);
    hipMemcpyAsync(&Res[start_idx], z.getCUDAResPtr()[i], nEle * sizeof(T),
                    hipMemcpyDeviceToHost);
    for (unsigned int j = 0; j < N; ++j)
      hipMemcpyAsync(&Grad[j][start_idx], &z.getCUDAGradPtr()[i][j * nEle],
                      nEle * sizeof(T), hipMemcpyDeviceToHost);
    start_idx += nEle;
  }
  s << "[Res: "
    << "[ ";
  for (std::size_t i = 0; i < nEle; ++i)
    s << Res[i] << ", ";
  s << "]," << std::endl;
  for (unsigned int i = 0; i < N; ++i) {
    s << "Grad[" << i << "]: "
      << "[ ";
    for (std::size_t j = 0; j < nEle; ++j)
      s << Grad[i][j] << ", ";
    s << "]," << std::endl;
  }
  s << "_device: " << std::to_string(z.getDevice()) << "]";
  return s;
}
template std::ostream &ostreamCUDA<float>(std::ostream &s,
                                          const JetVector<float> &);
template std::ostream &ostreamCUDA<double>(std::ostream &s,
                                           const JetVector<double> &);

template class JetVector<float>;
template class JetVector<double>;
}  // namespace MegBA
