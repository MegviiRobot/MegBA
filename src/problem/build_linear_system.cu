#include "hip/hip_runtime.h"
/**
* MegBA is Licensed under the Apache License, Version 2.0 (the "License")
*
* Copyright (c) 2021 Megvii Inc. All rights reserved.
*
**/

#include <thrust/transform.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>
#include <Eigen/Sparse>
#include "macro.h"
#include "edge/base_edge.h"
#include "wrapper.hpp"
#include "resource/handle_manager.h"

#if __CUDA_ARCH__ < 600 && defined(__CUDA_ARCH__)
union AtomicUnion{
    double dValue;
    unsigned long long ullValue;
};

__inline__ __device__ double atomicAdd(double* address, double val) {
    AtomicUnion old, assumed;
    old.dValue = *address;

    do {
        assumed = old;
        old.ullValue = atomicCAS(reinterpret_cast<unsigned long long *>(address),
                                  assumed.ullValue,
                                  AtomicUnion{val + assumed.dValue}.ullValue);

        // Note: uses integer comparison to
        // avoid hang in case of NaN (since NaN != NaN)
    } while (assumed.ullValue != old.ullValue);

    return old.dValue;
}
#endif

namespace MegBA {
namespace {
template <typename T>
__device__ void makeHpp(const T *valSmem, const T valI, const int cameraDim,
                        const int hppCsrRowI, T *hppCsrVal) {
  for (int i = 0; i < cameraDim; ++i)
    atomicAdd(&hppCsrVal[hppCsrRowI + i],
              valI * valSmem[i * blockDim.x + threadIdx.x]);
}

template <typename T>
__device__ void makeHpl(const T *valSmem, const T valI,
                        const int relativePositionPoint, const int pointDim,
                        const int cameraDim, int hplCsrRow, T *hplCsrVal) {
  const int hplCsrRowI = hplCsrRow + relativePositionPoint * pointDim;

  for (int i = 0; i < pointDim; ++i) {
    hplCsrVal[hplCsrRowI + i] +=
        valI * valSmem[(i + cameraDim) * blockDim.x + threadIdx.x];
  }
}

template <typename T>
__device__ void makeHlp(const T *valSmem, const T valI,
                        const int relativePositionCamera, const int cameraDim,
                        int hlpCsrRow, T *hlpCsrVal) {
  const int hlpCsrRow_i = hlpCsrRow + relativePositionCamera * cameraDim;

  for (int i = 0; i < cameraDim; ++i) {
    hlpCsrVal[hlpCsrRow_i + i] += valI * valSmem[i * blockDim.x + threadIdx.x];
  }
}

template <typename T>
__device__ void makeHll(const T *valSmem, const T valI, const int pointDim,
                        const int cameraDim, const int hllPosition,
                        T *hllMatrix) {
  for (int i = 0; i < pointDim; ++i)
    atomicAdd(&hllMatrix[hllPosition + i],
              valI * valSmem[(i + cameraDim) * blockDim.x + threadIdx.x]);
}

template <typename T>
__global__ void
makeHSchur(
    const T *const *const valPtrs, const T *const *const errorPtrs,
    const int *absolutePositionCamera, const int *absolutePositionPoint,
    const int *relativePositionCamera, const int *relativePositionPoint,
    const int *hplCsrRowPtr, const int *hlpCsrRowPtr, const int resDim,
    const int cameraDim, const int pointDim, const int errorNum, T *gCamera,
    T *gPoint, T *hppCsrVal, T *hllCsrVal, T *hplCsrVal, T *hlpCsrVal) {
  /*
                 * make sure that blockDim.x % 32 == 0, if so, there won't be any thread divergence within a wrap.
   */
  const unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= errorNum)
    return;

  T *valSmem = Wrapper::Shared_Memory<T>::get();

  const int absolutePositionPointLocal = absolutePositionPoint[tid];
  const int absolutePositionCameraLocal = absolutePositionCamera[tid];
  const int relativePositionPointLocal = relativePositionPoint[tid];
  const int relativePositionCameraLocal = relativePositionCamera[tid];

  T gSum{0.};
  for (int i = 0; i < resDim; ++i) {
    const T valI = valPtrs[i][errorNum * threadIdx.y + tid];
    __syncthreads();
    valSmem[threadIdx.y * blockDim.x + threadIdx.x] = valI;
    __syncthreads();

    if (threadIdx.y < cameraDim) {
      makeHpp(valSmem, valI, cameraDim,
              (absolutePositionCameraLocal * cameraDim + threadIdx.y) *
                  cameraDim,
              hppCsrVal);
      makeHpl(
          valSmem, valI, relativePositionPointLocal, pointDim, cameraDim,
          hplCsrRowPtr[absolutePositionCameraLocal * cameraDim + threadIdx.y],
          hplCsrVal);
    } else {
      makeHll(valSmem, valI, pointDim, cameraDim,
              absolutePositionPointLocal * (pointDim * pointDim) +
                  (threadIdx.y - cameraDim) * pointDim /* hllPosition */,
              hllCsrVal);
      makeHlp(valSmem, valI, relativePositionCameraLocal, cameraDim,
              hlpCsrRowPtr[absolutePositionPointLocal * pointDim + threadIdx.y -
                           cameraDim],
              hlpCsrVal);
    }
    gSum += -valI * errorPtrs[i][tid];
  }

  if (threadIdx.y < cameraDim) {
    atomicAdd(&gCamera[absolutePositionCameraLocal * cameraDim + threadIdx.y],
              gSum);
  } else {
    atomicAdd(&gPoint[absolutePositionPointLocal * pointDim + threadIdx.y -
                      cameraDim],
              gSum);
  }
}
}  // namespace

template <typename T, int result_weight = 1, int dest_weight = 0>
__global__ void oursGgemvBatched(const T *csrVal, const T *r,
                                 int batchSize,
                                 T *dx);

template <typename T>
void EdgeVector<T>::buildLinearSystemSchurCUDA(const JVD<T> &jetEstimation) {
  const auto rows = jetEstimation.rows(), cols = jetEstimation.cols();
  const auto cameraDim = edges[0].getGradShape();
  const auto pointDim = edges[1].getGradShape();
  const auto cameraNum = num[0];
  const auto pointNum = num[1];
  const auto hppRows = cameraDim * cameraNum;
  const auto hllRows = pointDim * pointNum;
  ASSERT_CUDA_NO_ERROR();

  std::vector<T *> gCameraDevice{static_cast<std::size_t>(_option.worldSize)};
  std::vector<T *> gPointDevice{static_cast<std::size_t>(_option.worldSize)};
  for (int i = 0; i < _option.worldSize; ++i) {
    hipSetDevice(i);
    hipMemsetAsync(schurEquationContainer[i].g, 0,
                    (hppRows + hllRows) * sizeof(T));
    gCameraDevice[i] = &schurEquationContainer[i].g[0];
    gPointDevice[i] = &schurEquationContainer[i].g[hppRows];
    ASSERT_CUDA_NO_ERROR();
    hipMemsetAsync(schurEquationContainer[i].csrVal[0], 0,
                    schurEquationContainer[i].nnz[0] * sizeof(T));
    hipMemsetAsync(schurEquationContainer[i].csrVal[1], 0,
                    schurEquationContainer[i].nnz[1] * sizeof(T));
    hipMemsetAsync(schurEquationContainer[i].csrVal[2], 0,
                    schurEquationContainer[i].nnz[2] * sizeof(T));
    hipMemsetAsync(schurEquationContainer[i].csrVal[3], 0,
                    schurEquationContainer[i].nnz[3] * sizeof(T));
    ASSERT_CUDA_NO_ERROR();
  }

  const auto resDim = rows * cols;
  std::vector<std::unique_ptr<const T *[]>> totalPtrs {};
  totalPtrs.reserve(_option.worldSize);
  std::vector<const T **> totalPtrsDevice{
      static_cast<std::size_t>(_option.worldSize)};

  std::vector<const T **> valPtrs{static_cast<std::size_t>(_option.worldSize)};
  std::vector<const T **> valPtrsDevice{
      static_cast<std::size_t>(_option.worldSize)};

  std::vector<const T **> errorPtrs{
      static_cast<std::size_t>(_option.worldSize)};
  std::vector<const T **> errorPtrsDevice{
      static_cast<std::size_t>(_option.worldSize)};
  for (int deviceRank = 0; deviceRank < _option.worldSize; ++deviceRank) {
    totalPtrs.emplace_back(new const T *[resDim * (3 + resDim)]);
    hipSetDevice(deviceRank);
    hipMalloc(&totalPtrsDevice[deviceRank],
               resDim * (3 + resDim) * sizeof(T *));

    valPtrs[deviceRank] = &totalPtrs[deviceRank][0];
    valPtrsDevice[deviceRank] = &totalPtrsDevice[deviceRank][0];

    errorPtrs[deviceRank] = &totalPtrs[deviceRank][resDim];
    errorPtrsDevice[deviceRank] = &totalPtrsDevice[deviceRank][resDim];
    for (int i = 0; i < rows; ++i)
      for (int j = 0; j < cols; ++j) {
        const auto &jetEstimationInner = jetEstimation(i, j);
        valPtrs[deviceRank][j + i * cols] =
            jetEstimationInner.getCUDAGradPtr()[deviceRank];
        errorPtrs[deviceRank][j + i * cols] =
            jetEstimationInner.getCUDAResPtr()[deviceRank];
      }
    hipMemcpyAsync(totalPtrsDevice[deviceRank], totalPtrs[deviceRank].get(),
                    resDim * 2 * sizeof(T *), hipMemcpyHostToDevice);
  }

  if (jetInformation.rows() != 0 && jetInformation.cols() != 0) {
    // TODO(Jie Ren): implement this
  } else {
    for (int i = 0; i < _option.worldSize; ++i) {
      hipSetDevice(i);
      const auto edgeNum = MemoryPool::getElmNum(i);
      dim3 block(std::min((decltype(edgeNum))32, edgeNum),
                 cameraDim + pointDim);
      dim3 grid((edgeNum - 1) / block.x + 1);
      makeHSchur<<<grid, block, block.x * block.y * sizeof(T)>>>(
          valPtrsDevice[i], errorPtrsDevice[i],
          schurPositionAndRelationContainer[i].absolutePositionCamera,
          schurPositionAndRelationContainer[i].absolutePositionPoint,
          schurPositionAndRelationContainer[i].relativePositionCamera,
          schurPositionAndRelationContainer[i].relativePositionPoint,
          schurEquationContainer[i].csrRowPtr[0],
          schurEquationContainer[i].csrRowPtr[1], resDim, cameraDim, pointDim,
          edgeNum, gCameraDevice[i], gPointDevice[i],
          schurEquationContainer[i].csrVal[2],
          schurEquationContainer[i].csrVal[3],
          schurEquationContainer[i].csrVal[0],
          schurEquationContainer[i].csrVal[1]);
    }
  }
  ASSERT_CUDA_NO_ERROR();
  for (int i = 0; i < _option.worldSize; ++i) {
    hipSetDevice(i);
    hipStreamSynchronize(nullptr);
    hipFree(totalPtrsDevice[i]);
  }

  const auto &comms = HandleManager::getNcclComm();
  ncclGroupStart();
  for (int i = 0; i < _option.worldSize; ++i) {
    ncclAllReduce(schurEquationContainer[i].csrVal[2],
                  schurEquationContainer[i].csrVal[2],
                  schurEquationContainer[i].nnz[2],
                  Wrapper::declared_cudaDatatype<T>::nccl_dtype, ncclSum,
                  comms[i], nullptr);
    ncclAllReduce(schurEquationContainer[i].csrVal[3],
                  schurEquationContainer[i].csrVal[3],
                  schurEquationContainer[i].nnz[3],
                  Wrapper::declared_cudaDatatype<T>::nccl_dtype, ncclSum,
                  comms[i], nullptr);
    ncclAllReduce(gCameraDevice[i], gCameraDevice[i], hppRows + hllRows,
                  Wrapper::declared_cudaDatatype<T>::nccl_dtype, ncclSum,
                  comms[i], nullptr);
  }
  ncclGroupEnd();
}

template class EdgeVector<double>;
template class EdgeVector<float>;
}  // namespace MegBA
