#include "hip/hip_runtime.h"
/**
* MegBA is Licensed under the Apache License, Version 2.0 (the "License")
*
* Copyright (c) 2021 Megvii Inc. All rights reserved.
*
**/

#include <edge/BaseEdge.h>
#include <Macro.h>

namespace MegBA {
template <typename T> void EdgeVector<T>::backupDaPtrs() {
  if (_option.use_schur) {
    const auto grad_shape = getGradShape();
    for (int i = 0; i < Memory_Pool::getWorldSize(); ++i) {
      hipSetDevice(i);
      hipMemcpyAsync(schurDaPtrsOld[0][i], schurDaPtrs[0][i],
                      Memory_Pool::getElmNum(i) * grad_shape * sizeof(T),
                      hipMemcpyDeviceToDevice, schurStreamLmMemcpy[i]);
    }
  } else {
    // TODO: implement this
  }
}

namespace {
template <typename T>
__global__ void BroadCastCsrColInd(const int *input, const int other_dim,
                                   const int nElm, int *output) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= nElm)
    return;
  for (int i = 0; i < other_dim; ++i) {
    output[i + tid * other_dim] = i + input[tid] * other_dim;
  }
}
}

    template <typename T> void EdgeVector<T>::preparePositionAndRelationDataCUDA() {

  if (_option.use_schur) {
    std::vector<std::array<int *, 2>> CompressedCsrColInd;
    CompressedCsrColInd.resize(_option.world_size);
    for (int i = 0; i < _option.world_size; ++i) {
      hipSetDevice(i);
      const auto edge_num = Memory_Pool::getElmNum(i);

      hipMalloc(&schurEquationContainer[i].csrRowPtr[0],
                 (num[0] * schurEquationContainer[i].dim[0] + 1) * sizeof(int));
      hipMalloc(&schurEquationContainer[i].csrRowPtr[1],
                 (num[1] * schurEquationContainer[i].dim[1] + 1) * sizeof(int));
      hipMemcpyAsync(
          schurEquationContainer[i].csrRowPtr[0], schurCsrRowPtr[i][0].get(),
          (num[0] * schurEquationContainer[i].dim[0] + 1) * sizeof(int),
          hipMemcpyHostToDevice);
      hipMemcpyAsync(
          schurEquationContainer[i].csrRowPtr[1], schurCsrRowPtr[i][1].get(),
          (num[1] * schurEquationContainer[i].dim[1] + 1) * sizeof(int),
          hipMemcpyHostToDevice);

      hipMalloc(&schurEquationContainer[i].csrVal[0],
                 schurEquationContainer[i].nnz[0] * sizeof(T)); // hpl
      hipMalloc(&schurEquationContainer[i].csrColInd[0],
                 schurEquationContainer[i].nnz[0] * sizeof(int));
      {
        const std::size_t entries_in_rows =
            schurEquationContainer[i].nnz[0] / schurEquationContainer[i].dim[1];
        dim3 block(std::min(entries_in_rows, (std::size_t)512));
        dim3 grid((entries_in_rows - 1) / block.x + 1);
        hipMalloc(&CompressedCsrColInd[i][0], entries_in_rows * sizeof(int));
        hipMemcpyAsync(CompressedCsrColInd[i][0],
                        schurHEntrance[i].csrColInd_[0].get(),
                        entries_in_rows * sizeof(int), hipMemcpyHostToDevice);
        ASSERT_CUDA_NO_ERROR();
        BroadCastCsrColInd<T><<<grid, block>>>(
            CompressedCsrColInd[i][0], schurEquationContainer[i].dim[1],
            entries_in_rows, schurEquationContainer[i].csrColInd[0]);
        ASSERT_CUDA_NO_ERROR();
      }

      hipMalloc(&schurEquationContainer[i].csrVal[1],
                 schurEquationContainer[i].nnz[1] * sizeof(T)); // hlp
      hipMalloc(&schurEquationContainer[i].csrColInd[1],
                 schurEquationContainer[i].nnz[1] * sizeof(int));
      {
        const std::size_t entries_in_rows =
            schurEquationContainer[i].nnz[1] / schurEquationContainer[i].dim[0];
        dim3 block(std::min(entries_in_rows, (std::size_t)512));
        dim3 grid((entries_in_rows - 1) / block.x + 1);
        hipMalloc(&CompressedCsrColInd[i][1], entries_in_rows * sizeof(int));
        hipMemcpyAsync(CompressedCsrColInd[i][1],
                        schurHEntrance[i].csrColInd_[1].get(),
                        entries_in_rows * sizeof(int), hipMemcpyHostToDevice);
        ASSERT_CUDA_NO_ERROR();
        BroadCastCsrColInd<T><<<grid, block>>>(
            CompressedCsrColInd[i][1], schurEquationContainer[i].dim[0],
            entries_in_rows, schurEquationContainer[i].csrColInd[1]);
        ASSERT_CUDA_NO_ERROR();
      }

      hipMalloc(&schurEquationContainer[i].csrVal[2],
                 schurEquationContainer[i].nnz[2] * sizeof(T)); // hpp

      hipMalloc(&schurEquationContainer[i].csrVal[3],
                 schurEquationContainer[i].nnz[3] * sizeof(T)); // hll

      hipMalloc(&schurEquationContainer[i].g,
                 (num[0] * schurEquationContainer[i].dim[0] +
                  num[1] * schurEquationContainer[i].dim[1]) *
                     sizeof(T));

      hipMalloc(&schurPositionAndRelationContainer[i].relativePositionCamera,
                 edge_num * sizeof(int));
      hipMemcpyAsync(
          schurPositionAndRelationContainer[i].relativePositionCamera,
          schurRelativePosition[0][i].data(), edge_num * sizeof(int),
          hipMemcpyHostToDevice);

      hipMalloc(&schurPositionAndRelationContainer[i].relativePositionPoint,
                 edge_num * sizeof(int));
      hipMemcpyAsync(
          schurPositionAndRelationContainer[i].relativePositionPoint,
          schurRelativePosition[1][i].data(), edge_num * sizeof(int),
          hipMemcpyHostToDevice);

      hipMalloc(&schurPositionAndRelationContainer[i].absolutePositionCamera,
                 cameraVertexNum * edge_num * sizeof(int));
      hipMemcpyAsync(
          schurPositionAndRelationContainer[i].absolutePositionCamera,
          schurAbsolutePosition[0][i].data(), edge_num * sizeof(int),
          hipMemcpyHostToDevice);

      hipMalloc(&schurPositionAndRelationContainer[i].absolutePositionPoint,
                 pointVertexNum * edge_num * sizeof(int));
      hipMemcpyAsync(
          schurPositionAndRelationContainer[i].absolutePositionPoint,
          schurAbsolutePosition[1][i].data(), edge_num * sizeof(int),
          hipMemcpyHostToDevice);
    }
    for (int i = 0; i < _option.world_size; ++i) {
      hipSetDevice(i);
      hipDeviceSynchronize();
      hipFree(CompressedCsrColInd[i][0]);
      hipFree(CompressedCsrColInd[i][1]);
    }
  } else {
    // TODO: implement this
  }
}

template <typename T> void EdgeVector<T>::cudaPrepareUpdateData() {
  if (_option.use_schur) {
    const auto world_size = Memory_Pool::getWorldSize();
    const auto grad_shape = getGradShape();
    schurStreamLmMemcpy.resize(world_size);
    std::vector<T *> da_ptrs_, da_ptrs_old_;
    da_ptrs_.resize(world_size);
    da_ptrs_old_.resize(world_size);
    for (int i = 0; i < world_size; ++i) {
      hipSetDevice(i);
      hipStreamCreateWithFlags(&schurStreamLmMemcpy[i],
                                hipStreamNonBlocking);
      T *da_ptr, *da_ptr_old;
      const auto nElm = Memory_Pool::getElmNum(i);
      hipMalloc(&da_ptr, nElm * grad_shape * sizeof(T));
      hipMalloc(&da_ptr_old, nElm * grad_shape * sizeof(T));
      da_ptrs_[i] = da_ptr;
      da_ptrs_old_[i] = da_ptr_old;
    }
    schurDaPtrs.resize(cameraVertexNum + pointVertexNum);
    schurDaPtrsOld.resize(cameraVertexNum + pointVertexNum);
    for (int i = 0; i < cameraVertexNum + pointVertexNum; ++i) {
      schurDaPtrs[i].resize(world_size);
      schurDaPtrsOld[i].resize(world_size);
    }
    for (int i = 0, i_unfixed = 0, offset = 0; i < edges.size(); ++i) {
      if (edges[i][0]->get_Fixed())
        continue;
      for (int j = 0; j < world_size; ++j) {
        const auto nElm = Memory_Pool::getElmNum(j);
        schurDaPtrs[i_unfixed][j] = &da_ptrs_[j][offset * nElm];
        schurDaPtrsOld[i_unfixed][j] = &da_ptrs_old_[j][offset * nElm];
      }
      i_unfixed++;
      const auto &estimation = edges[i][0]->get_Estimation();
      offset += estimation.rows() * estimation.cols();
    }
  } else {
    // TODO: implement this
  }
}

template <typename T> void EdgeVector<T>::deallocateResourceCUDA() {
  if (_option.use_schur) {
    for (int i = 0; i < Memory_Pool::getWorldSize(); ++i) {
      hipSetDevice(i);
      schurPositionAndRelationContainer[i].clearCUDA();
      hipFree(schurDaPtrs[0][i]);
      hipFree(schurDaPtrs[1][i]);
      hipFree(schurDaPtrsOld[0][i]);
      hipFree(schurDaPtrsOld[1][i]);
      hipStreamDestroy(schurStreamLmMemcpy[i]);
    }

    for (auto &edge : edges)
      edge.CPU();
  } else {
    // TODO: implement this
  }
}

template <typename T> void EdgeVector<T>::SchurEquationContainer::clearCUDA() {
  for (int i = 0; i < 2; ++i)
    hipFree(csrRowPtr[i]);
  for (int i = 0; i < 4; ++i)
    hipFree(csrVal[i]);
  for (int i = 0; i < 2; ++i)
    hipFree(csrColInd[i]);
  hipFree(g);
}

template <typename T>
void EdgeVector<T>::PositionAndRelationContainer::clearCUDA() {
  hipFree(relativePositionCamera);
  hipFree(relativePositionPoint);
  hipFree(absolutePositionCamera);
  hipFree(absolutePositionPoint);
  hipFree(connectionNumPoint);
}

template class EdgeVector<float>;
template class EdgeVector<double>;
}