#include "hip/hip_runtime.h"
/**
* MegBA is Licensed under the Apache License, Version 2.0 (the "License")
*
* Copyright (c) 2021 Megvii Inc. All rights reserved.
*
**/

#include "edge/base_edge.h"

namespace MegBA {
template <typename T> void EdgeVector<T>::backupValueDevicePtrs() const {
  if (option.useSchur) {
    const auto gradShape = getGradShape();
    for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
      hipSetDevice(i);
      hipMemcpyAsync(schurValueDevicePtrsOld[0][i], schurValueDevicePtrs[0][i],
                      MemoryPool::getItemNum(i) * gradShape * sizeof(T),
                      hipMemcpyDeviceToDevice, schurStreamLmMemcpy[i]);
    }
  } else {
    // TODO(Jie Ren): implement this
  }
}

namespace {
template <typename T>
__global__ void broadCastCsrColInd(const int *input, const int other_dim,
                                   const int nItem, int *output) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= nItem)
    return;
  for (int i = 0; i < other_dim; ++i) {
    output[i + tid * other_dim] = i + input[tid] * other_dim;
  }
}
}  // namespace

template <typename T> void EdgeVector<T>::preparePositionAndRelationDataCUDA() {
//  if (option.useSchur) {
//    std::vector<std::array<int *, 2>> compressedCsrColInd;
//    compressedCsrColInd.resize(option.deviceUsed.size());
//    for (int i = 0; i < option.deviceUsed.size(); ++i) {
//      hipSetDevice(i);
//      const auto edgeNum = MemoryPool::getItemNum(i);
//
//      hipMalloc(&schurEquationContainer[i].csrRowPtr[0],
//                 (num[0] * schurEquationContainer[i].dim[0] + 1) * sizeof(int));
//      hipMalloc(&schurEquationContainer[i].csrRowPtr[1],
//                 (num[1] * schurEquationContainer[i].dim[1] + 1) * sizeof(int));
//      hipMemcpyAsync(
//          schurEquationContainer[i].csrRowPtr[0], schurCsrRowPtr[i][0].get(),
//          (num[0] * schurEquationContainer[i].dim[0] + 1) * sizeof(int),
//          hipMemcpyHostToDevice);
//      hipMemcpyAsync(
//          schurEquationContainer[i].csrRowPtr[1], schurCsrRowPtr[i][1].get(),
//          (num[1] * schurEquationContainer[i].dim[1] + 1) * sizeof(int),
//          hipMemcpyHostToDevice);
//
//      hipMalloc(&schurEquationContainer[i].csrVal[0],
//                 schurEquationContainer[i].nnz[0] * sizeof(T));  // hpl
//      hipMalloc(&schurEquationContainer[i].csrColInd[0],
//                 schurEquationContainer[i].nnz[0] * sizeof(int));
//      {
//        const std::size_t entriesInRows =
//            schurEquationContainer[i].nnz[0] / schurEquationContainer[i].dim[1];
//        dim3 block(std::min(entriesInRows, (std::size_t)512));
//        dim3 grid((entriesInRows - 1) / block.x + 1);
//        hipMalloc(&compressedCsrColInd[i][0], entriesInRows * sizeof(int));
//        hipMemcpyAsync(compressedCsrColInd[i][0],
//                        schurHessianEntrance[i].csrColInd[0].get(),
//                        entriesInRows * sizeof(int), hipMemcpyHostToDevice);
//        broadCastCsrColInd<T><<<grid, block>>>(
//            compressedCsrColInd[i][0], schurEquationContainer[i].dim[1],
//            entriesInRows, schurEquationContainer[i].csrColInd[0]);
//      }
//
//      hipMalloc(&schurEquationContainer[i].csrVal[1],
//                 schurEquationContainer[i].nnz[1] * sizeof(T));  // hlp
//      hipMalloc(&schurEquationContainer[i].csrColInd[1],
//                 schurEquationContainer[i].nnz[1] * sizeof(int));
//      {
//        const std::size_t entriesInRows =
//            schurEquationContainer[i].nnz[1] / schurEquationContainer[i].dim[0];
//        dim3 block(std::min(entriesInRows, (std::size_t)512));
//        dim3 grid((entriesInRows - 1) / block.x + 1);
//        hipMalloc(&compressedCsrColInd[i][1], entriesInRows * sizeof(int));
//        hipMemcpyAsync(compressedCsrColInd[i][1],
//                        schurHessianEntrance[i].csrColInd[1].get(),
//                        entriesInRows * sizeof(int), hipMemcpyHostToDevice);
//        broadCastCsrColInd<T><<<grid, block>>>(
//            compressedCsrColInd[i][1], schurEquationContainer[i].dim[0],
//            entriesInRows, schurEquationContainer[i].csrColInd[1]);
//      }
//
//      hipMalloc(&schurEquationContainer[i].csrVal[2],
//                 schurEquationContainer[i].nnz[2] * sizeof(T));  // hpp
//
//      hipMalloc(&schurEquationContainer[i].csrVal[3],
//                 schurEquationContainer[i].nnz[3] * sizeof(T));  // hll
//
//      hipMalloc(&schurEquationContainer[i].g,
//                 (num[0] * schurEquationContainer[i].dim[0] +
//                  num[1] * schurEquationContainer[i].dim[1]) *
//                     sizeof(T));
//
//      hipMalloc(&schurPositionAndRelationContainer[i].relativePositionCamera,
//                 edgeNum * sizeof(int));
//      hipMemcpyAsync(
//          schurPositionAndRelationContainer[i].relativePositionCamera,
//          schurRelativePosition[0][i].data(), edgeNum * sizeof(int),
//          hipMemcpyHostToDevice);
//
//      hipMalloc(&schurPositionAndRelationContainer[i].relativePositionPoint,
//                 edgeNum * sizeof(int));
//      hipMemcpyAsync(
//          schurPositionAndRelationContainer[i].relativePositionPoint,
//          schurRelativePosition[1][i].data(), edgeNum * sizeof(int),
//          hipMemcpyHostToDevice);
//
//      hipMalloc(&schurPositionAndRelationContainer[i].absolutePositionCamera,
//                 cameraVertexNum * edgeNum * sizeof(int));
//      hipMemcpyAsync(
//          schurPositionAndRelationContainer[i].absolutePositionCamera,
//          schurAbsolutePosition[0][i].data(), edgeNum * sizeof(int),
//          hipMemcpyHostToDevice);
//
//      hipMalloc(&schurPositionAndRelationContainer[i].absolutePositionPoint,
//                 pointVertexNum * edgeNum * sizeof(int));
//      hipMemcpyAsync(
//          schurPositionAndRelationContainer[i].absolutePositionPoint,
//          schurAbsolutePosition[1][i].data(), edgeNum * sizeof(int),
//          hipMemcpyHostToDevice);
//    }
//    for (int i = 0; i < option.deviceUsed.size(); ++i) {
//      hipSetDevice(i);
//      hipDeviceSynchronize();
//      hipFree(compressedCsrColInd[i][0]);
//      hipFree(compressedCsrColInd[i][1]);
//    }
//  } else {
//    // TODO(Jie Ren): implement this
//  }
}

template <typename T> void EdgeVector<T>::PrepareUpdateDataCUDA() {
  if (option.useSchur) {
    const auto worldSize = MemoryPool::getWorldSize();
    const auto gradShape = getGradShape();
    schurStreamLmMemcpy.resize(worldSize);
    std::vector<T *> valueDevicePtrs, valueDevicePtrsOld;
    valueDevicePtrs.resize(worldSize);
    valueDevicePtrsOld.resize(worldSize);
    for (int i = 0; i < worldSize; ++i) {
      hipSetDevice(i);
      hipStreamCreateWithFlags(&schurStreamLmMemcpy[i],
                                hipStreamNonBlocking);
      T *valueDevicePtr, *valueDevicePtrOld;
      const auto nItem = MemoryPool::getItemNum(i);
      hipMalloc(&valueDevicePtr, nItem * gradShape * sizeof(T));
      hipMalloc(&valueDevicePtrOld, nItem * gradShape * sizeof(T));
      valueDevicePtrs[i] = valueDevicePtr;
      valueDevicePtrsOld[i] = valueDevicePtrOld;
    }
    schurValueDevicePtrs.resize(cameraVertexNum + pointVertexNum);
    schurValueDevicePtrsOld.resize(cameraVertexNum + pointVertexNum);
    for (int i = 0; i < cameraVertexNum + pointVertexNum; ++i) {
      schurValueDevicePtrs[i].resize(worldSize);
      schurValueDevicePtrsOld[i].resize(worldSize);
    }
    for (int i = 0, iUnfixed = 0, offset = 0; i < edges.size(); ++i) {
      if (edges[i][0]->fixed)
        continue;
      for (int j = 0; j < worldSize; ++j) {
        const auto nItem = MemoryPool::getItemNum(j);
        schurValueDevicePtrs[iUnfixed][j] = &valueDevicePtrs[j][offset * nItem];
        schurValueDevicePtrsOld[iUnfixed][j] = &valueDevicePtrsOld[j][offset * nItem];
      }
      iUnfixed++;
      const auto &estimation = edges[i][0]->getEstimation();
      offset += estimation.rows() * estimation.cols();
    }
  } else {
    // TODO(Jie Ren): implement this
  }
}

template <typename T> void EdgeVector<T>::deallocateResourceCUDA() {
  if (option.useSchur) {
    for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
      hipSetDevice(i);
      schurPositionAndRelationContainer[i].clearCUDA();
      hipFree(schurValueDevicePtrs[0][i]);
      hipFree(schurValueDevicePtrs[1][i]);
      hipFree(schurValueDevicePtrsOld[0][i]);
      hipFree(schurValueDevicePtrsOld[1][i]);
      hipStreamDestroy(schurStreamLmMemcpy[i]);
    }

    for (auto &edge : edges)
      edge.CPU();
  } else {
    // TODO(Jie Ren): implement this
  }
}

template <typename T> void EdgeVector<T>::SchurEquationContainer::clearCUDA() {
  for (int i = 0; i < 2; ++i)
    hipFree(csrRowPtr[i]);
  for (int i = 0; i < 4; ++i)
    hipFree(csrVal[i]);
  for (int i = 0; i < 2; ++i)
    hipFree(csrColInd[i]);
  hipFree(g);
}

template <typename T>
void EdgeVector<T>::PositionAndRelationContainer::clearCUDA() {
  hipFree(relativePositionCamera);
  hipFree(relativePositionPoint);
  hipFree(absolutePositionCamera);
  hipFree(absolutePositionPoint);
}

template class EdgeVector<float>;
template class EdgeVector<double>;
}  // namespace MegBA
