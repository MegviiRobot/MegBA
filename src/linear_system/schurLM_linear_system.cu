#include "hip/hip_runtime.h"
/**
* MegBA is Licensed under the Apache License, Version 2.0 (the "License")
*
* Copyright (c) 2021 Megvii Inc. All rights reserved.
*
**/

#include "linear_system/schurLM_linear_system.h"
#include "wrapper.hpp"

namespace MegBA {
namespace {
void CUDART_CB freeCallback(void *ptr) { free(ptr); }

template <typename T>
__global__ void broadCastCsrColInd(const int *input, const int other_dim,
                                   const int nItem, int *output) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= nItem) return;
  for (int i = 0; i < other_dim; ++i) {
    output[i + tid * other_dim] = i + input[tid] * other_dim;
  }
}

template <typename T>
__global__ void weightedPlusKernel(int nItem, const T *x, const T *y, T weight,
                                   T *z) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= nItem) return;
  z[tid] = x[tid] + weight * y[tid];
}

template <typename T>
__global__ void fillPtr(const T *aData, T *ainvData, const int batchSize,
                        const int hRowsNumPow2, const T **a, T **ainv) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= batchSize) return;
  a[tid] = &aData[tid * hRowsNumPow2];
  ainv[tid] = &ainvData[tid * hRowsNumPow2];
}

template <typename T>
void invert(const T *aFlat, int n, const int num, T *cFlat) {
  hipblasHandle_t handle = HandleManager::getCUBLASHandle()[0];

  const T **a;
  T **ainv;
  hipMalloc(&a, num * sizeof(T *));
  hipMalloc(&ainv, num * sizeof(T *));
  dim3 blockDim(std::min(decltype(num)(256), num));
  dim3 gridDim((num - 1) / blockDim.x + 1);

  fillPtr<<<gridDim, blockDim>>>(aFlat, cFlat, num, n * n, a, ainv);

  int *info;
  hipMalloc(&info, num * sizeof(int));
  Wrapper::cublasGmatinvBatched::call(handle, n, a, n, ainv, n, info, num);

  hipDeviceSynchronize();

  hipFree(a);
  hipFree(ainv);
  hipFree(info);
}

template <typename T>
void invertDistributed(const std::vector<T *> &aFlat, int n, const int num,
                       std::vector<T *> &cFlat) {
  const auto &handle = HandleManager::getCUBLASHandle();
  const auto worldSize = MemoryPool::getWorldSize();

  std::vector<const T **> a{static_cast<std::size_t>(worldSize)};
  std::vector<T **> ainv{static_cast<std::size_t>(worldSize)};
  std::vector<int *> info{static_cast<std::size_t>(worldSize)};
  dim3 blockDim(std::min(decltype(num)(256), num));
  dim3 gridDim((num - 1) / blockDim.x + 1);

  for (int i = 0; i < worldSize; ++i) {
    MemoryPool::allocateNormal(reinterpret_cast<void **>(&a[i]),
                               num * sizeof(T *), i);
    MemoryPool::allocateNormal(reinterpret_cast<void **>(&ainv[i]),
                               num * sizeof(T *), i);
    MemoryPool::allocateNormal(reinterpret_cast<void **>(&info[i]),
                               num * sizeof(int), i);
  }

  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    fillPtr<<<gridDim, blockDim>>>(aFlat[i], cFlat[i], num, n * n, a[i],
                                   ainv[i]);

    Wrapper::cublasGmatinvBatched::call(handle[i], n, a[i], n, ainv[i], n,
                                        info[i], num);
  }

  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    hipDeviceSynchronize();
    MemoryPool::deallocateNormal(info[i], i);
    MemoryPool::deallocateNormal(ainv[i], i);
    MemoryPool::deallocateNormal(a[i], i);
  }
}

template <typename T, int result_weight = 1, int dest_weight = 0>
__global__ void oursGgemvBatched(const T *csrVal, const T *r, int batchSize,
                                 T *dx) {
  /*
blockDim, x-dim: camera or point dim, y-dim: process how many cameras/points in
this block
   */
  unsigned int tid = threadIdx.y + blockIdx.x * blockDim.y;
  if (tid >= batchSize) return;

  T *smem = Wrapper::SharedMemory<T>::get();
  T sum = 0;
  smem[threadIdx.x + threadIdx.y * blockDim.x] =
      r[threadIdx.x + tid * blockDim.x];
  __syncthreads();
  for (unsigned int i = 0; i < blockDim.x; ++i) {
    sum +=
        csrVal[i + threadIdx.x * blockDim.x + tid * blockDim.x * blockDim.x] *
        smem[i + threadIdx.y * blockDim.x];
  }
  dx[threadIdx.x + tid * blockDim.x] =
      result_weight * sum + dest_weight * dx[threadIdx.x + tid * blockDim.x];
}

template <typename T>
bool schurPCGSolverDistributedCUDA(
    const std::vector<T *> &SpMVbuffer, SolverOption::SolverOptionPCG option,
    const int cameraNum, const int pointNum, const int cameraDim,
    const int pointDim, const std::vector<int> &hplNnz, const int hppRows,
    const int hllRows, const std::vector<T *> &hppCsrVal,
    const std::vector<T *> &hplCsrVal, const std::vector<int *> &hplCsrColInd,
    const std::vector<int *> &hplCsrRowPtr, const std::vector<T *> &hlpCsrVal,
    const std::vector<int *> &hlpCsrColInd,
    const std::vector<int *> &hlpCsrRowPtr,
    const std::vector<T *> &hllInvCsrVal, const std::vector<T *> &g,
    const std::vector<T *> &d_x) {
  const auto &comms = HandleManager::getNCCLComm();
  const auto worldSize = MemoryPool::getWorldSize();
  constexpr auto hipDataType = Wrapper::declaredDtype<T>::cudaDtype;
  const auto &cusparseHandle = HandleManager::getCUSPARSEHandle();
  const auto &cublasHandle = HandleManager::getCUBLASHandle();
  std::vector<hipStream_t> cusparseStream, cublasStream;
  const T one{1.0}, zero{0.0}, neg_one{-1.0};
  T alphaN, alphaNegN, rhoNm1;
  std::vector<T> dot;
  std::vector<T *> hppInvCsrVal, pN, rN, axN, temp, deltaXBackup;
  std::vector<hipsparseSpMatDescr_t> hpl, hlp;
  std::vector<hipsparseDnVecDescr_t> vecx, vecp, vecAx, vectemp;
  cusparseStream.resize(worldSize);
  cublasStream.resize(worldSize);
  dot.resize(worldSize);
  hppInvCsrVal.resize(worldSize);
  pN.resize(worldSize);
  rN.resize(worldSize);
  axN.resize(worldSize);
  temp.resize(worldSize);
  deltaXBackup.resize(worldSize);
  hpl.resize(worldSize);
  hlp.resize(worldSize);
  vecx.resize(worldSize);
  vecp.resize(worldSize);
  vecAx.resize(worldSize);
  vectemp.resize(worldSize);
  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    hipsparseGetStream(cusparseHandle[i], &cusparseStream[i]);
    hipblasGetStream(cublasHandle[i], &cublasStream[i]);
    MemoryPool::allocateNormal(reinterpret_cast<void **>(&hppInvCsrVal[i]),
                               hppRows * cameraDim * sizeof(T), i);
    MemoryPool::allocateNormal(reinterpret_cast<void **>(&pN[i]),
                               hppRows * sizeof(T), i);
    MemoryPool::allocateNormal(reinterpret_cast<void **>(&rN[i]),
                               hppRows * sizeof(T), i);
    MemoryPool::allocateNormal(reinterpret_cast<void **>(&axN[i]),
                               hppRows * sizeof(T), i);
    MemoryPool::allocateNormal(reinterpret_cast<void **>(&temp[i]),
                               hllRows * sizeof(T), i);

    MemoryPool::allocateNormal(reinterpret_cast<void **>(&deltaXBackup[i]),
                               hllRows * sizeof(T), i);

    hipMemcpyAsync(rN[i], g[i], hppRows * sizeof(T), hipMemcpyDeviceToDevice);

    /* Wrap raw data into cuSPARSE generic API objects */
    hipsparseCreateCsr(&hpl[i], hppRows, hllRows, hplNnz[i], hplCsrRowPtr[i],
                      hplCsrColInd[i], hplCsrVal[i], HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
                      hipDataType);
    hipsparseCreateCsr(&hlp[i], hllRows, hppRows, hplNnz[i], hlpCsrRowPtr[i],
                      hlpCsrColInd[i], hlpCsrVal[i], HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
                      hipDataType);
    hipsparseCreateDnVec(&vecx[i], hppRows, d_x[i], hipDataType);
    hipsparseCreateDnVec(&vecp[i], hppRows, pN[i], hipDataType);
    hipsparseCreateDnVec(&vecAx[i], hppRows, axN[i], hipDataType);
    hipsparseCreateDnVec(&vectemp[i], hllRows, temp[i], hipDataType);
  }

  invertDistributed(hppCsrVal, cameraDim, cameraNum, hppInvCsrVal);

  /* Allocate workspace for cuSPARSE */
  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    /* Begin CG */
    // x1 = ET*x
    hipsparseSpMV(cusparseHandle[i], HIPSPARSE_OPERATION_NON_TRANSPOSE, &one,
                 hlp[i], vecx[i], &zero, vectemp[i], hipDataType,
                 HIPSPARSE_SPMV_ALG_DEFAULT, SpMVbuffer[i]);
  }

  ncclGroupStart();
  for (int i = 0; i < worldSize; ++i) {
    ncclAllReduce(temp[i], temp[i], hllRows,
                  Wrapper::declaredDtype<T>::ncclDtype, ncclSum,
                  comms[i], cusparseStream[i]);
  }
  ncclGroupEnd();

  for (int i = 0; i < worldSize; ++i) {
    dim3 block(pointDim, std::min(32, pointNum));
    dim3 grid((pointNum - 1) / block.y + 1);
    hipSetDevice(i);
    // borrow pN as temp workspace
    oursGgemvBatched<<<grid, block, block.x * block.y * sizeof(T),
                       cusparseStream[i]>>>(hllInvCsrVal[i], temp[i], pointNum,
                                            temp[i]);

    hipsparseSpMV(cusparseHandle[i], HIPSPARSE_OPERATION_NON_TRANSPOSE, &one,
                 hpl[i], vectemp[i], &zero, vecAx[i], hipDataType,
                 HIPSPARSE_SPMV_ALG_DEFAULT, SpMVbuffer[i]);
  }

  ncclGroupStart();
  for (int i = 0; i < worldSize; ++i) {
    ncclAllReduce(axN[i], axN[i], hppRows,
                  Wrapper::declaredDtype<T>::ncclDtype, ncclSum,
                  comms[i], cusparseStream[i]);
  }
  ncclGroupEnd();

  for (int i = 0; i < worldSize; ++i) {
    dim3 block(cameraDim, std::min(32, cameraNum));
    dim3 grid((cameraNum - 1) / block.y + 1);
    hipSetDevice(i);
    oursGgemvBatched<T, 1, -1>
        <<<grid, block, block.x * block.y * sizeof(T), cusparseStream[i]>>>(
            hppCsrVal[i], d_x[i], cameraNum, axN[i]);
  }
  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    hipStreamSynchronize(cusparseStream[i]);
    // r = b - Ax
    Wrapper::cublasGaxpy::call(cublasHandle[i], hppRows, &neg_one, axN[i], 1,
                               rN[i], 1);
  }
  int n{0};
  T rhoN{0};
  T rhoMinimum = INFINITY;
  std::vector<T> rho_n_item;
  rho_n_item.resize(worldSize);
  bool done{false};
  do {
    std::size_t offset{0};
    rhoN = 0;
    for (int i = 0; i < worldSize; ++i) {
      dim3 block(cameraDim, std::min(32, cameraNum));
      dim3 grid((cameraNum - 1) / block.y + 1);
      hipSetDevice(i);
      // borrow axN
      oursGgemvBatched<<<grid, block, block.x * block.y * sizeof(T),
                         cublasStream[i]>>>(hppInvCsrVal[i], rN[i], cameraNum,
                                            axN[i]);

      // rhoN = rTr
      const auto nItem = MemoryPool::getItemNum(i, hppRows);
      Wrapper::cublasGdot::call(cublasHandle[i], nItem, &rN[i][offset], 1,
                                &axN[i][offset], 1, &rho_n_item[i]);
      offset += nItem;
    }
    for (int i = 0; i < worldSize; ++i) {
      hipSetDevice(i);
      hipStreamSynchronize(cublasStream[i]);
      rhoN += rho_n_item[i];
    }
    if (rhoN > option.refuseRatio * rhoMinimum) {
      for (int i = 0; i < worldSize; ++i) {
        hipSetDevice(i);
        hipMemcpyAsync(d_x[i], deltaXBackup[i], hppRows * sizeof(T),
                        hipMemcpyDeviceToDevice);
      }
      break;
    }
    rhoMinimum = std::min(rhoMinimum, rhoN);

    if (n >= 1) {
      T beta_n = rhoN / rhoNm1;
      for (int i = 0; i < worldSize; ++i) {
        dim3 block(std::min(256, hppRows));
        dim3 grid((hppRows - 1) / block.x + 1);
        hipSetDevice(i);
        weightedPlusKernel<T>
            <<<grid, block>>>(hppRows, axN[i], pN[i], beta_n, pN[i]);
      }
    } else {
      for (int i = 0; i < worldSize; ++i) {
        hipSetDevice(i);
        Wrapper::cublasGcopy::call(cublasHandle[i], hppRows, axN[i], 1, pN[i],
                                   1);
      }
    }

    for (int i = 0; i < worldSize; ++i) {
      // Ax = Ad ???? q = Ad
      // x1 = ET*x
      hipSetDevice(i);
      hipStreamSynchronize(cublasStream[i]);
      hipsparseSpMV(cusparseHandle[i], HIPSPARSE_OPERATION_NON_TRANSPOSE, &one,
                   hlp[i], vecp[i], &zero, vectemp[i], hipDataType,
                   HIPSPARSE_SPMV_ALG_DEFAULT, SpMVbuffer[i]);
    }

    ncclGroupStart();
    for (int i = 0; i < worldSize; ++i) {
      ncclAllReduce(temp[i], temp[i], hllRows,
                    Wrapper::declaredDtype<T>::ncclDtype, ncclSum,
                    comms[i], cusparseStream[i]);
    }
    ncclGroupEnd();

    for (int i = 0; i < worldSize; ++i) {
      dim3 block(pointDim, std::min(32, pointNum));
      dim3 grid((pointNum - 1) / block.y + 1);
      hipSetDevice(i);
      // borrow pN as temp workspace
      oursGgemvBatched<<<grid, block, block.x * block.y * sizeof(T),
                         cusparseStream[i]>>>(hllInvCsrVal[i], temp[i],
                                              pointNum, temp[i]);

      hipsparseSpMV(cusparseHandle[i], HIPSPARSE_OPERATION_NON_TRANSPOSE, &one,
                   hpl[i], vectemp[i], &zero, vecAx[i], hipDataType,
                   HIPSPARSE_SPMV_ALG_DEFAULT, SpMVbuffer[i]);
    }

    ncclGroupStart();
    for (int i = 0; i < worldSize; ++i) {
      ncclAllReduce(axN[i], axN[i], hppRows,
                    Wrapper::declaredDtype<T>::ncclDtype, ncclSum,
                    comms[i], cusparseStream[i]);
    }
    ncclGroupEnd();

    for (int i = 0; i < worldSize; ++i) {
      dim3 block(cameraDim, std::min(32, cameraNum));
      dim3 grid((cameraNum - 1) / block.y + 1);
      hipSetDevice(i);
      oursGgemvBatched<T, 1, -1>
          <<<grid, block, block.x * block.y * sizeof(T), cusparseStream[i]>>>(
              hppCsrVal[i], pN[i], cameraNum, axN[i]);
    }

    offset = 0;
    for (int i = 0; i < worldSize; ++i) {
      hipSetDevice(i);
      hipStreamSynchronize(cusparseStream[i]);
      // dot :dTq
      const auto nItem = MemoryPool::getItemNum(i, hppRows);
      Wrapper::cublasGdot::call(cublasHandle[i], nItem, &pN[i][offset], 1,
                                &axN[i][offset], 1, &dot[i]);
      offset += nItem;
    }
    // beta_n: one = rhoN / dTq
    double dot_sum{0};
    for (int i = 0; i < worldSize; ++i) {
      hipSetDevice(i);
      hipStreamSynchronize(cublasStream[i]);
      dot_sum += dot[i];
    }
    alphaN = rhoN / dot_sum;
    for (int i = 0; i < worldSize; ++i) {
      hipSetDevice(i);
      // x=x+alphaN*pN
      hipMemcpyAsync(deltaXBackup[i], d_x[i], hppRows * sizeof(T),
                      hipMemcpyDeviceToDevice);
      Wrapper::cublasGaxpy::call(cublasHandle[i], hppRows, &alphaN, pN[i], 1,
                                 d_x[i], 1);
    }

    alphaNegN = -alphaN;

    for (int i = 0; i < worldSize; ++i) {
      hipSetDevice(i);
      // r = r - alphaN*Ax = r - alphaN*q
      Wrapper::cublasGaxpy::call(cublasHandle[i], hppRows, &alphaNegN, axN[i],
                                 1, rN[i], 1);
    }
    rhoNm1 = rhoN;
    // printf("iteration = %3d, residual = %f\n", n, std::abs(rhoN));
    ++n;
    done = std::abs(rhoN) < option.tol;
  } while (!done && n < option.maxIter);
  // hipSetDevice(0);
  // PRINT_DMEMORY_SEGMENT(d_x[0], 0, 2, T);
  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    hipsparseDestroySpMat(hpl[i]);
    hipsparseDestroySpMat(hlp[i]);
    hipsparseDestroyDnVec(vecx[i]);
    hipsparseDestroyDnVec(vecAx[i]);
    hipsparseDestroyDnVec(vecp[i]);
    hipsparseDestroyDnVec(vectemp[i]);

    MemoryPool::deallocateNormal(deltaXBackup[i], i);
    MemoryPool::deallocateNormal(temp[i], i);
    MemoryPool::deallocateNormal(axN[i], i);
    MemoryPool::deallocateNormal(rN[i], i);
    MemoryPool::deallocateNormal(pN[i], i);
    MemoryPool::deallocateNormal(hppInvCsrVal[i], i);
  }
  return done;
}

template <typename T>
void schurMakeVDistributed(std::vector<T *> *SpMVbuffer, const int pointNum,
                           const int pointDim, const std::vector<int> &hplNnz,
                           const int hppRows, const int hllRows,
                           const std::vector<T *> &hplCsrVal,
                           const std::vector<int *> &hplCsrColInd,
                           const std::vector<int *> &hplCsrRowPtr,
                           const std::vector<T *> &hllInvCsrVal,
                           const std::vector<T *> &r) {
  const auto &comms = HandleManager::getNCCLComm();
  const auto worldSize = MemoryPool::getWorldSize();
  const auto &cusparseHandle = HandleManager::getCUSPARSEHandle();
  constexpr auto hipDataType = Wrapper::declaredDtype<T>::cudaDtype;

  std::vector<T *> v, w;
  std::vector<hipStream_t> cusparseStream;
  std::vector<hipsparseDnVecDescr_t> vecv, vecw;
  std::vector<hipsparseSpMatDescr_t> hpl;
  v.resize(worldSize);
  w.resize(worldSize);
  cusparseStream.resize(worldSize);
  vecv.resize(worldSize);
  vecw.resize(worldSize);
  hpl.resize(worldSize);
  for (int i = 0; i < worldSize; ++i) {
    hipsparseGetStream(cusparseHandle[i], &cusparseStream[i]);
    v[i] = &r[i][0];
    w[i] = &r[i][hppRows];
    hipsparseCreateDnVec(&vecv[i], hppRows, v[i], hipDataType);
    hipsparseCreateDnVec(&vecw[i], hllRows, w[i], hipDataType);
    hipsparseCreateCsr(&hpl[i], hppRows, hllRows, hplNnz[i], hplCsrRowPtr[i],
                      hplCsrColInd[i], hplCsrVal[i], HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
                      hipDataType);
  }

  dim3 blockDim(pointDim, std::min(32, pointNum));
  dim3 gridDim((pointNum - 1) / blockDim.y + 1);
  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    // notably, w here is changed(w = C^{-1}w),
    // so later w = C^{-1}(w - ETv) = C^{-1}w - C^{-1}ETv -> w = w - C^{-1}ETv
    oursGgemvBatched<<<gridDim, blockDim, blockDim.x * blockDim.y * sizeof(T),
                       cusparseStream[i]>>>(hllInvCsrVal[i], w[i], pointNum,
                                            w[i]);
  }

  T alpha{-1.0}, beta = T(1. / worldSize);

  SpMVbuffer->resize(worldSize);
  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    size_t bufferSize = 0;
    hipsparseSpMV_bufferSize(cusparseHandle[i], HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha, hpl[i], vecw[i], &beta, vecv[i],
                            hipDataType, HIPSPARSE_SPMV_ALG_DEFAULT,
                            &bufferSize);
    MemoryPool::allocateNormal(
        reinterpret_cast<void **>(&SpMVbuffer->operator[](i)), bufferSize, i);
    hipsparseSpMV(cusparseHandle[i], HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                 hpl[i], vecw[i], &beta, vecv[i], hipDataType,
                 HIPSPARSE_SPMV_ALG_DEFAULT, SpMVbuffer->operator[](i));
  }

  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    hipStreamSynchronize(cusparseStream[i]);

    hipsparseDestroySpMat(hpl[i]);
    hipsparseDestroyDnVec(vecv[i]);
    hipsparseDestroyDnVec(vecw[i]);
  }
  ncclGroupStart();
  for (int i = 0; i < worldSize; ++i) {
    ncclAllReduce(v[i], v[i], hppRows,
                  Wrapper::declaredDtype<T>::ncclDtype, ncclSum,
                  comms[i], cusparseStream[i]);
  }
  ncclGroupEnd();
}

template <typename T>
void schurSolveWDistributed(
    const std::vector<T *> &SpMVbuffer, const int pointNum, const int pointDim,
    const std::vector<int> &hplNnz, const int hppRows, const int hllRows,
    const std::vector<T *> &hlpCsrVal, const std::vector<int *> &hlpCsrColInd,
    const std::vector<int *> &hlpCsrRowPtr,
    const std::vector<T *> &hllInvCsrVal, const std::vector<T *> &d_r,
    const std::vector<T *> &d_x) {
  const auto comms = HandleManager::getNCCLComm();
  const auto worldSize = MemoryPool::getWorldSize();
  constexpr auto hipDataType = Wrapper::declaredDtype<T>::cudaDtype;

  std::vector<T *> xc, xp, w;
  xc.resize(worldSize);
  xp.resize(worldSize);
  w.resize(worldSize);
  for (int i = 0; i < worldSize; ++i) {
    xc[i] = &d_x[i][0];
    xp[i] = &d_x[i][hppRows];
    w[i] = &d_r[i][hppRows];
  }

  const auto &cusparseHandle = HandleManager::getCUSPARSEHandle();

  std::vector<hipStream_t> cusparseStream;
  std::vector<hipsparseDnVecDescr_t> vecxc, vecxp, vecw;
  std::vector<hipsparseSpMatDescr_t> hlp;
  cusparseStream.resize(worldSize);
  vecxc.resize(worldSize);
  vecxp.resize(worldSize);
  vecw.resize(worldSize);
  hlp.resize(worldSize);

  for (int i = 0; i < worldSize; ++i) {
    hipsparseGetStream(cusparseHandle[i], &cusparseStream[i]);

    hipsparseCreateDnVec(&vecxc[i], hppRows, xc[i], hipDataType);
    hipsparseCreateDnVec(&vecxp[i], hllRows, xp[i], hipDataType);
    hipsparseCreateDnVec(&vecw[i], hllRows, w[i], hipDataType);
    hipsparseCreateCsr(&hlp[i], hllRows, hppRows, hplNnz[i], hlpCsrRowPtr[i],
                      hlpCsrColInd[i], hlpCsrVal[i], HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
                      hipDataType);
  }

  T alpha{1.0}, beta{0.0};
  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    // x1 = ET*x
    hipsparseSpMV(cusparseHandle[i], HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                 hlp[i], vecxc[i], &beta, vecxp[i], hipDataType,
                 HIPSPARSE_SPMV_ALG_DEFAULT, SpMVbuffer[i]);
  }

  ncclGroupStart();
  for (int i = 0; i < worldSize; ++i) {
    ncclAllReduce(xp[i], xp[i], hllRows,
                  Wrapper::declaredDtype<T>::ncclDtype, ncclSum,
                  comms[i], cusparseStream[i]);
  }
  ncclGroupEnd();

  dim3 blockDim(pointDim, std::min(32, pointNum));
  dim3 gridDim((pointNum - 1) / blockDim.y + 1);
  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    oursGgemvBatched<T, -1, 1>
        <<<gridDim, blockDim, blockDim.x * blockDim.y * sizeof(T),
           cusparseStream[i]>>>(hllInvCsrVal[i], xp[i], pointNum, w[i]);
    hipMemcpyAsync(xp[i], w[i], hllRows * sizeof(T), hipMemcpyDeviceToDevice,
                    cusparseStream[i]);
  }

  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    hipStreamSynchronize(cusparseStream[i]);

    hipsparseDestroySpMat(hlp[i]);
    hipsparseDestroyDnVec(vecxc[i]);
    hipsparseDestroyDnVec(vecw[i]);
  }
}

template <typename T>
bool SchurPCGSolverDistributed(
    const SolverOption::SolverOptionPCG &option,
    const std::vector<T *> &hppCsrVal, const std::vector<T *> &hllCsrVal,
    const std::vector<T *> &hplCsrVal, const std::vector<int *> &hplCsrColInd,
    const std::vector<int *> &hplCsrRowPtr, const std::vector<T *> &hlpCsrVal,
    const std::vector<int *> &hlpCsrColInd,
    const std::vector<int *> &hlpCsrRowPtr, const std::vector<T *> &g,
    int cameraDim, int cameraNum, int pointDim, int pointNum,
    const std::vector<int> &hplNnz, int hppRows, int hllRows,
    const std::vector<T *> &deltaX) {
  // hll inverse-----------------------------------------------------------
  const auto worldSize = MemoryPool::getWorldSize();

  std::vector<T *> SpMVbuffer;

  std::vector<T *> hllInvCsrVal;
  hllInvCsrVal.resize(worldSize);
  for (int i = 0; i < worldSize; ++i) {
    MemoryPool::allocateNormal(reinterpret_cast<void **>(&hllInvCsrVal[i]),
                               hllRows * pointDim * sizeof(T), i);
  }
  invertDistributed(hllCsrVal, pointDim, pointNum, hllInvCsrVal);

  schurMakeVDistributed(&SpMVbuffer, pointNum, pointDim, hplNnz, hppRows,
                        hllRows, hplCsrVal, hplCsrColInd, hplCsrRowPtr,
                        hllInvCsrVal, g);
  bool PCG_success = schurPCGSolverDistributedCUDA(
      SpMVbuffer, option, cameraNum, pointNum, cameraDim, pointDim, hplNnz,
      hppRows, hllRows, hppCsrVal, hplCsrVal, hplCsrColInd, hplCsrRowPtr,
      hlpCsrVal, hlpCsrColInd, hlpCsrRowPtr, hllInvCsrVal, g, deltaX);
  schurSolveWDistributed(SpMVbuffer, pointNum, pointDim, hplNnz, hppRows,
                         hllRows, hlpCsrVal, hlpCsrColInd, hlpCsrRowPtr,
                         hllInvCsrVal, g, deltaX);
  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    hipDeviceSynchronize();
    MemoryPool::deallocateNormal(SpMVbuffer[i], i);
    MemoryPool::deallocateNormal(hllInvCsrVal[i], i);
  }
  return PCG_success;
}
}  // namespace

template <typename T>
void SchurLMLinearSystem<T>::allocateResourceCUDA() {
  const auto worldSize = MemoryPool::getWorldSize();
  std::vector<std::array<int *, 2>> compressedCsrColInd;
  compressedCsrColInd.resize(worldSize);
  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    hipMalloc(&this->deltaXPtrBackup[i], this->getHessianShape() * sizeof(T));
    hipMalloc(&this->deltaXPtr[i], this->getHessianShape() * sizeof(T));
    hipMemsetAsync(this->deltaXPtr[i], 0, this->getHessianShape() * sizeof(T));

    hipMalloc(&this->extractedDiag[i][0], this->dim[0] * this->num[0] * sizeof(T));
    hipMalloc(&this->extractedDiag[i][1], this->dim[1] * this->num[1] * sizeof(T));

    std::array<int *, 2> csrRowPtrHost{equationContainers[i].csrRowPtr};
    hipMalloc(&equationContainers[i].csrRowPtr[0],
               (this->num[0] * this->dim[0] + 1) * sizeof(int));
    hipMalloc(&equationContainers[i].csrRowPtr[1],
               (this->num[1] * this->dim[1] + 1) * sizeof(int));
    hipMemcpyAsync(equationContainers[i].csrRowPtr[0], csrRowPtrHost[0],
                    (this->num[0] * this->dim[0] + 1) * sizeof(int),
                    hipMemcpyHostToDevice);
    hipLaunchHostFunc(nullptr, freeCallback, (void *)csrRowPtrHost[0]);
    hipMemcpyAsync(equationContainers[i].csrRowPtr[1], csrRowPtrHost[1],
                    (this->num[1] * this->dim[1] + 1) * sizeof(int),
                    hipMemcpyHostToDevice);
    hipLaunchHostFunc(nullptr, freeCallback, (void *)csrRowPtrHost[1]);

    std::array<int *, 2> csrColIndHost{equationContainers[i].csrColInd};
    hipMalloc(&equationContainers[i].csrVal[0],
               equationContainers[i].nnz[0] * sizeof(T));  // hpl
    hipMalloc(&equationContainers[i].csrColInd[0],
               equationContainers[i].nnz[0] * sizeof(int));
    {
      const std::size_t entriesInRows = equationContainers[i].nnz[0] / this->dim[1];
      dim3 block(std::min(entriesInRows, (std::size_t)512));
      dim3 grid((entriesInRows - 1) / block.x + 1);
      hipMalloc(&compressedCsrColInd[i][0], entriesInRows * sizeof(int));
      hipMemcpyAsync(compressedCsrColInd[i][0], csrColIndHost[0],
                      entriesInRows * sizeof(int), hipMemcpyHostToDevice);
      hipLaunchHostFunc(nullptr, freeCallback, (void *)csrColIndHost[0]);
      broadCastCsrColInd<T>
          <<<grid, block>>>(compressedCsrColInd[i][0], this->dim[1], entriesInRows,
                            equationContainers[i].csrColInd[0]);
    }

    hipMalloc(&equationContainers[i].csrVal[1],
               equationContainers[i].nnz[1] * sizeof(T));  // hlp
    hipMalloc(&equationContainers[i].csrColInd[1],
               equationContainers[i].nnz[1] * sizeof(int));
    {
      const std::size_t entriesInRows = equationContainers[i].nnz[1] / this->dim[0];
      dim3 block(std::min(entriesInRows, (std::size_t)512));
      dim3 grid((entriesInRows - 1) / block.x + 1);
      hipMalloc(&compressedCsrColInd[i][1], entriesInRows * sizeof(int));
      hipMemcpyAsync(compressedCsrColInd[i][1], csrColIndHost[1],
                      entriesInRows * sizeof(int), hipMemcpyHostToDevice);
      hipLaunchHostFunc(nullptr, freeCallback, (void *)csrColIndHost[1]);
      broadCastCsrColInd<T>
          <<<grid, block>>>(compressedCsrColInd[i][1], this->dim[0], entriesInRows,
                            equationContainers[i].csrColInd[1]);
    }

    hipMalloc(&equationContainers[i].csrVal[2],
               equationContainers[i].nnz[2] * sizeof(T));  // hpp

    hipMalloc(&equationContainers[i].csrVal[3],
               equationContainers[i].nnz[3] * sizeof(T));  // hll

    hipMalloc(&this->g[i], this->getHessianShape() * sizeof(T));
    hipMalloc(&this->gBackup[i], this->getHessianShape() * sizeof(T));
  }
  for (int i = 0; i < worldSize; ++i) {
    hipSetDevice(i);
    hipDeviceSynchronize();
    hipFree(compressedCsrColInd[i][0]);
    hipFree(compressedCsrColInd[i][1]);
  }
}

namespace {
template <typename T>
__global__ void RecoverDiagKernel(const T *in, const T a, const int batchSize,
                                  T *out) {
  /*
   * blockDim, x-dim: camera or point dim, y-dim: process how many cameras/points in this block
   */
  unsigned int tid = threadIdx.y + blockIdx.x * blockDim.y;
  if (tid >= batchSize) return;

  out[threadIdx.x + threadIdx.x * blockDim.x + tid * blockDim.x * blockDim.x] =
      (a + 1) * in[threadIdx.x + tid * blockDim.x];
}

template <typename T>
void RecoverDiag(const T *diag, const T a, const int batchSize, const int dim,
                 T *csrVal) {
  dim3 block(dim, std::min(decltype(batchSize)(32), batchSize));
  dim3 grid((batchSize - 1) / block.y + 1);
  RecoverDiagKernel<T><<<grid, block>>>(diag, a, batchSize, csrVal);
}

template <typename T>
__global__ void ExtractOldAndApplyNewDiagKernel(const T a, const int batchSize,
                                                T *csrVal, T *diags) {
  /*
   * blockDim, x-dim: camera or point dim, y-dim: process how many cameras/points in this block
   */
  unsigned int tid = threadIdx.y + blockIdx.x * blockDim.y;
  if (tid >= batchSize) return;

  const T diag = csrVal[threadIdx.x + threadIdx.x * blockDim.x +
                        tid * blockDim.x * blockDim.x];
  diags[threadIdx.x + tid * blockDim.x] = diag;
  csrVal[threadIdx.x + threadIdx.x * blockDim.x +
         tid * blockDim.x * blockDim.x] = (a + 1) * diag;
}

template <typename T>
void extractOldAndApplyNewDiag(const T a, const int batchSize, const int dim,
                               T *csrVal, T *diag) {
  dim3 block(dim, std::min(decltype(batchSize)(32), batchSize));
  dim3 grid((batchSize - 1) / block.y + 1);
  ExtractOldAndApplyNewDiagKernel<<<grid, block>>>(a, batchSize, csrVal, diag);
}
}

template <typename T>
void SchurLMLinearSystem<T>::processDiag(
    const AlgoStatus::AlgoStatusLM &lmAlgoStatus) const {
  if (lmAlgoStatus.recoverDiag) {
    for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
      hipSetDevice(i);
      auto &container = equationContainers[i];
      RecoverDiag(this->extractedDiag[i][0], T(1. / lmAlgoStatus.region), this->num[0],
                  this->dim[0], container.csrVal[2]);
      RecoverDiag(this->extractedDiag[i][1], T(1. / lmAlgoStatus.region), this->num[1],
                  this->dim[1], container.csrVal[3]);
    }
  } else {
    for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
      hipSetDevice(i);
      auto &container = equationContainers[i];
      extractOldAndApplyNewDiag(T(1. / lmAlgoStatus.region), this->num[0], this->dim[0],
                                container.csrVal[2], this->extractedDiag[i][0]);
      extractOldAndApplyNewDiag(T(1. / lmAlgoStatus.region), this->num[1], this->dim[1],
                                container.csrVal[3], this->extractedDiag[i][1]);
    }
  }
}

template <typename T>
void SchurLMLinearSystem<T>::backup() const {
  const int hessianShape = this->getHessianShape();
  for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
    hipSetDevice(i);
    hipMemcpyAsync(this->deltaXPtrBackup[i], this->deltaXPtr[i],
                    hessianShape * sizeof(T), hipMemcpyDeviceToDevice);
    hipMemcpyAsync(this->gBackup[i], this->g[i],
                    hessianShape * sizeof(T), hipMemcpyDeviceToDevice);
  }
}

template <typename T>
void SchurLMLinearSystem<T>::rollback() const {
  const int hessianShape = this->getHessianShape();
  for (int i = 0; i < MemoryPool::getWorldSize(); ++i) {
    hipSetDevice(i);
    hipMemcpyAsync(this->deltaXPtr[i], this->deltaXPtrBackup[i],
                    hessianShape * sizeof(T), hipMemcpyDeviceToDevice);
    hipMemcpyAsync(this->g[i], this->gBackup[i],
                    hessianShape * sizeof(T), hipMemcpyDeviceToDevice);
  }
}

template <typename T>
void SchurLMLinearSystem<T>::solve() const {
  const std::size_t worldSize = MemoryPool::getWorldSize();
  std::vector<T *> hppCsrVal{worldSize};
  std::vector<T *> hllCsrVal{worldSize};
  std::vector<T *> hplCsrVal{worldSize};
  std::vector<T *> hlpCsrVal{worldSize};
  std::vector<int *> hplCsrColInd{worldSize};
  std::vector<int *> hlpCsrColInd{worldSize};
  std::vector<int *> hplCsrRowPtr{worldSize};
  std::vector<int *> hlpCsrRowPtr{worldSize};
  std::vector<T *> g{worldSize};
  std::vector<int> hplNnz{};
  hplNnz.resize(worldSize);
  std::vector<T *> deltaX{worldSize};

  for (int i = 0; i < worldSize; ++i) {
    hppCsrVal[i] = equationContainers[i].csrVal[2];
    hllCsrVal[i] = equationContainers[i].csrVal[3];
    hplCsrVal[i] = equationContainers[i].csrVal[0];
    hlpCsrVal[i] = equationContainers[i].csrVal[1];
    hplCsrColInd[i] = equationContainers[i].csrColInd[0];
    hlpCsrColInd[i] = equationContainers[i].csrColInd[1];
    hplCsrRowPtr[i] = equationContainers[i].csrRowPtr[0];
    hlpCsrRowPtr[i] = equationContainers[i].csrRowPtr[1];
    g[i] = this->g[i];
    hplNnz[i] = equationContainers[i].nnz[0];
    deltaX[i] = this->deltaXPtr[i];
  }

  SchurPCGSolverDistributed(this->solverOption.solverOptionPCG, hppCsrVal,
                            hllCsrVal, hplCsrVal, hplCsrColInd, hplCsrRowPtr,
                            hlpCsrVal, hlpCsrColInd, hlpCsrRowPtr, g, this->dim[0],
                            this->num[0], this->dim[1], this->num[1], hplNnz, this->dim[0] * this->num[0],
                            this->dim[1] * this->num[1], deltaX);
}

template <typename T>
void SchurLMLinearSystem<T>::applyUpdate(T *xPtr) const {
  const auto &cublasHandle = HandleManager::getCUBLASHandle();
  const T one = 1.;
  hipSetDevice(0);
  Wrapper::cublasGaxpy::call(cublasHandle[0], this->getHessianShape(), &one,
                             this->deltaXPtr[0], 1, xPtr, 1);
}

template struct SchurLMLinearSystem<double>;
template struct SchurLMLinearSystem<float>;
}
