/**
* MegBA is Licensed under the Apache License, Version 2.0 (the "License")
*
* Copyright (c) 2021 Megvii Inc. All rights reserved.
*
**/

#include "linear_system/base_linear_system.h"

namespace MegBA {
template <typename T>
void BaseLinearSystem<T>::freeCUDA() {
  for (int i = 0; i < problemOption.deviceUsed.size(); ++i) {
    hipSetDevice(i);
    hipFree(deltaXPtr[i]);
    hipFree(g[i]);
  }
}

template class BaseLinearSystem<double>;
template class BaseLinearSystem<float>;
}
